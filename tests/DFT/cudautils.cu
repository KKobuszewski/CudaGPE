#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cudautils.cuh"


extern hipEvent_t start_t;
extern hipEvent_t stop_t;

double print_cudatimeit(const char* message) {
  float computationTime;
  hipEventElapsedTime(&computationTime, start_t, stop_t);
  printf( "time of %-60s %lf s\n", message, (double) computationTime );
  return (double) computationTime;
}

double fprint_cudatimeit(FILE* file) {
  float computationTime;
  hipEventElapsedTime(&computationTime, start_t, stop_t);
  fwrite(&computationTime, (size_t) sizeof(float), 1, file);
  return (double) computationTime;
}

// void cudaCheckErrors(hipError_t err,const char* action){
//   if (hipGetLastError() != hipSuccess)
//   {
//         fprintf(stderr, "Failed to:<< %s >>(error code %s)!\n", action, hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//   }
// }