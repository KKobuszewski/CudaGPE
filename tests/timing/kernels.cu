#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "kernels.h"

#define BLOCK_SIZE 1024 // max 1024 <- it can be 3d like 1024x1x1, 512x2x1, 256x2x2 etc. etc.
#define GRID_SIZE 1

/*
 * compile: nvcc -o prog saxpy.cu
 */

/*
 * Simple kernel adding a vector scaled by constant to another vector (overwriting second vec)
 */
__global__
void saxpy(int n, const double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] += a*x[i];
  
}

/*
 * Funtion to checking errors from CUDA functions.
 * hipError_t err - structure with error code, hipSuccess means no error
 * char* action - description of action (to be easier to find in code)
 */
void cudaCheckErrors(hipError_t err,const char* action){
  if (err != hipSuccess)
  {
        fprintf(stderr, "Failed to:<< %s >>(error code %s)!\n", action, hipGetErrorString(err));
        exit(EXIT_FAILURE);
  }
}

/*
 * This function provides interface to CUDA
 * return status of action (if there was a crash)
 */
void perform_cuda_kernel(int N, const double a, double *x, double *y) {
  
  printf("\n\n\nGPU\n");
  
  double *x_dev, *y_dev;
  
  // GPU MEM
  clock_t temp_t = clock();
  clock_t start = clock();
  hipMalloc(&x_dev, N*sizeof(double)); 
  hipMalloc(&y_dev, N*sizeof(double));
  clock_t gpu_mem_t = clock() - start;
  
  start = clock();
  hipMemcpy(x_dev, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_dev, y, N*sizeof(double), hipMemcpyHostToDevice);
  clock_t gpu_cpy_t = clock() - start;
  printf("\nMemory allocation on DEVICE:\t\t\t\t %lf s\n", gpu_mem_t/((double)CLOCKS_PER_SEC));
  printf("Memory copying time on DEVICE:\t\t\t\t %lf s\n", gpu_cpy_t/((double)CLOCKS_PER_SEC));
  printf("Copying + memalloc on DEVICE:\t\t\t\t %lf s\n", (gpu_cpy_t+gpu_mem_t)/((double)CLOCKS_PER_SEC));
  
  
  
  
  
  // GPU COMPUTING
  printf("\n<<%d, %d>>, N=%d\n",GRID_SIZE, BLOCK_SIZE,N);
  
  // for timing purposes
  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  // start timer
  hipEventRecord(start_event,0);
  
  //saxpy<<<(N+255)/256, 256>>>(N, a, x_dev, y_dev); <- tu cos nie dziala tak jak trzeba
  for (int ii = 0; ii<FOR_LOOPS; ii++) { // FOR_LOOPS defined in kernels.h
    saxpy<<<GRID_SIZE, BLOCK_SIZE>>>(N, a, x_dev, y_dev);
    hipDeviceSynchronize();
  }
  
  hipEventRecord(stop_event,0);
  hipEventSynchronize(stop_event);
  float computationTime;
  hipEventElapsedTime(&computationTime, start_event, stop_event);
  
  cudaCheckErrors(hipGetLastError(),"cuda kernel");
  
  
  printf("\nGPU operation time:\t\t\t\t\t %f s\n", computationTime/1000.0);
  
  
  
  
  // copying data back to host
  start = clock();
  
  cudaCheckErrors(
    hipMemcpy(y, y_dev, N*sizeof(double), hipMemcpyDeviceToHost),
  "copying data back to host");
  
  clock_t gpu_copyback_t = clock() - start;
  printf("\nCopying data from GPU back to host:\t\t\t %lf s\n", gpu_copyback_t/((double)CLOCKS_PER_SEC));
  
  
  
  
  // FREE MEM ON DEVICE
  start = clock();
  
  cudaCheckErrors(	hipFree(x_dev)	,"free x_dev vec");
  cudaCheckErrors(	hipFree(y_dev)	,"free y_dev vec");
  
  clock_t gpu_cufree_t = clock() - start;
  printf("\nFreeing memory on DEVICE:\t\t\t\t %lf s\n", gpu_cufree_t/((double)CLOCKS_PER_SEC));
  //printf("\nTotal time on DEVICE:\t\t\t\t %lf s\n", (clock() - temp_t)/((double)CLOCKS_PER_SEC));
  
}