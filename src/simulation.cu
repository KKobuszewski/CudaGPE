#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <pthread.h>
#include <math.h>
#include <complex.h>
//#include <gsl/...> <- NAJLEPIEJ STWORZYC LINK DO TEJ BIBLIOTEKI I DODAC DO /usr/include, /usr/lib/ BO SIE ZLE LINKUJE!
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "global.h"
#include "dipolar.h"
#include "simulation.cuh"
#include "cudautils.cuh"
#include "kernels.cuh"


// global variables
extern Globals* global_stuff;
bool FLAG_RUN_SIMULATION = true;
extern const char* thread_names[];
extern const char* stream_names[];


// timing
hipEvent_t start_t;
hipEvent_t stop_t;

#ifdef IMAG_TIME
const uint64_t time_tot = 1e-02/DT;
//const uint64_t time_tot = 1000000;
#else
//const uint64_t time_tot = llround(0.318309886183791/DT); // no Vext revival time
const uint64_t time_tot = 2*llround((2*3.14159265358979323846/OMEGA)/DT); // harmonic potential revival time
#endif
const uint64_t frames_to_be_saved = 100;


// pthread managment
pthread_barrier_t barrier;


// cuda libs global variables
hipblasHandle_t cublas_handle;
hipfftHandle* cufft_plans;


// pointers to device functions
//dev_funcZ_ptr_t operator_T_h_ptr;
//dev_funcZ_ptr_t operator_Vext_h_ptr;

// variables on device only
//__constant__ double* norm_dev_con; // <- copy this variable to constant memory


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							FUNC DEFINITIONS								 *
 * 																	 *
 * ************************************************************************************************************************************* */


void free_device();
void save_stats(uint64_t step_index);
void save_simulation_params();


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							COMPUTIONAL FUNCTIONS								 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * This function counts chemical potential in time-independent Gross-Pitaevskii equation from relative norm change during ITE step.
 * ( Wavefunction is being normed every step, so the relative change is the value of norm in current step! )
 * 
 */
static inline long double chemical_potential_ite(long double norm) {
    return -logl(norm)/( (long double) DT);
}
/*
 * Less accurate but faster version of function above.
 */
static inline double chemical_potential_ite(double norm) {
    return -log(norm)/DT;
}


// interaction potential
static inline double interaction_potential(uint32_t ii) {
    return Vdd(kx(ii),1.,1.) + G_CONTACT; // dipolar + contact interactions
}



/* ************************************************************************************************************************************* *
 * 																	 *
 * 							SIM VARIABLES									 *
 * 																	 *
 * ************************************************************************************************************************************* */

// data structures on host (pinnable memory) TODO: Check how much of this could be allocated!
double complex* wf_host;
double complex* propagator_T_host;
double complex* propagator_Vext_host;
double complex* density_k_host;
double* density_r_host;
double* Vdd_host;

// data structures on device
hipDoubleComplex* complex_arr1_dev; // pointer on array holding wavefunction in device memory
hipDoubleComplex* complex_arr2_dev;
hipDoubleComplex* complex_arr3_dev;
double* double_arr1_dev;
hipDoubleComplex* propagator_T_dev; // array of constant factors e^-ik**2/2dt
hipDoubleComplex* propagator_Vext_dev; // array of constant factors e^-iVextdt
double* Vdd_dev; // array of costant factors <- count on host with spec funcs lib or use Abramowitz & Stegun approximation

// statistics
double norm_host;
double* norm_dev;

double chemical_potential_host;
double* chemical_potential_dev;

double mean_T_host;
double complex meanZ_T_host;
hipDoubleComplex* meanZ_T_dev;

double mean_Vext_host;
double complex meanZ_Vext_host;
hipDoubleComplex* meanZ_Vext_dev;

double mean_Vcon_host;
double complex meanZ_Vcon_host;
hipDoubleComplex* meanZ_Vcon_dev;

double mean_Vdip_host;
double complex meanZ_Vdip_host;
hipDoubleComplex* meanZ_Vdip_dev;

/* ************************************************************************************************************************************* *
 * 																	 *
 * 							SIM THREAD									 *
 * 																	 *
 * ************************************************************************************************************************************* *
 *
 * - allocation memory on host
 * - initialization of data
 * - main algorithm
 */
void* simulation_thread(void* passing_ptr) {
  
  
  
  //stick_this_thread_to_core(1); <- in cudautils, not used, include to header first
  pthread_barrier_wait (&barrier_global);
  printf("running %s thread.\n",thread_names[SIMULATION_THRD]);
  
  // allocate pinnable memory on host
  // arrays
  hipHostAlloc((void**) &(wf_host), sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  hipHostAlloc((void**) &propagator_T_host, sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc
  hipHostAlloc((void**) &propagator_Vext_host, sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  hipHostAlloc((void**) &density_k_host, sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // for copying denstiy of wf in Fourier space
  hipHostAlloc((void**) &density_r_host, sizeof(double)*NX*NY*NZ, hipHostMallocDefault);
  hipHostAlloc((void**) &Vdd_host, sizeof(double)*NX*NY*NZ, hipHostMallocDefault);
  
  //scalars
  hipHostAlloc((void**) &norm_host, sizeof(double), hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  
  printf("allocated memory on host.\n");
  
  
#ifdef DEBUG
  printf("1st barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
#endif
  pthread_barrier_wait (&barrier);
  // copy data async from host to device (if needed)
  if (global_stuff->init_wf_fd != -1) {
    // copy data from host to device (if needed) / cannot async because
    printf("copying initial wavefunction on device\n");
    HANDLE_ERROR( hipMemcpy(complex_arr1_dev, init_wf_mmap, NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) ); // change to asynchronous!
  }
  else {
    
    printf("initating wavefunction on device.\n");
#ifndef V_CON
    call_kernel_Z_1d( ker_gauss_1d, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
    //call_kernel_Z_1d( ker_const_1d, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
#else
    call_kernel_Z_1d( ker_gauss_1d, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
    //call_kernel_Z_1d( ker_const_1d, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
#endif    
  }
  
  // copying device pointers to host memory (to make them callable in kernels)
  //hipMemcpyFromSymbol(&operator_T_h_ptr, HIP_SYMBOL(operator_T_dev_ptr), sizeof(dev_funcZ_ptr_t));
  //hipMemcpyFromSymbol(&operator_Vext_h_ptr, HIP_SYMBOL(operator_Vext_dev_ptr), sizeof(dev_funcZ_ptr_t));
  
  printf("creating propagators\n");
  /*
   * TODO: - do it parallel with OpenMP! (6 cores)
   * 
   * it is possioble to do it on device
   * call_kernel_Z_1d( ker_create_propagator_T, propagator_T_dev, (streams)[HELPER_STREAM] );
   */
  omp_set_num_threads(6);
  #pragma omp parallel for
  for( uint64_t ii=0; ii < NX; ii++ ) {
#ifdef REAL_TIME
    propagator_T_host[ii] = cexp(-I*kx(ii)*(0.5*kx(ii)*DT));
    propagator_Vext_host[ii] = cexp(-I*(0.5*OMEGA)*(OMEGA*(ii*DX+XMIN))*((ii*DX+XMIN)*DT)); // <- !!! KOLEJNOSC MNOZEMIA A DOKLADNOSC !!!
#endif
#ifdef IMAG_TIME   
    propagator_T_host[ii] = cexpl(-kx(ii)*0.5*kx(ii)*DT);
    propagator_Vext_host[ii] = cexpl(-(0.5*OMEGA*OMEGA*(ii*DX+XMIN)*(ii*DX+XMIN)*DT)); // <- !!! KOLEJNOSC MNOZEMIA A DOKLADNOSC !!!
#endif
    //printf("%.15f + %.15fj\n",creal(propagator_Vext_host[ii]), cimag(propagator_Vext_host[ii]) );
#ifdef V_DIP
    // test it using contact interactions only
    Vdd_host[ii] = Vdd(kx(ii),1.,1.) + G_CONTACT*SQRT_2PI; // dipolar + contact interactions
#endif
  }
  
  /*
   * CHECK IF IT IS DONE IN pipelining?
   * probably not...
   */
  
  // copying propag T to dev
  HANDLE_ERROR( hipMemcpyAsync(propagator_T_dev, propagator_T_host,
				NX*NY*NZ*sizeof(hipDoubleComplex),
				hipMemcpyHostToDevice,
				(streams)[HELPER_STREAM]) );
  
  // copying after initialization (in meantime on another stream)
  HANDLE_ERROR( hipMemcpyAsync(wf_host, complex_arr1_dev,
				NX*NY*NZ*sizeof(hipDoubleComplex),
				hipMemcpyDeviceToHost,
				(streams)[SIMULATION_STREAM]) );
  hipDeviceSynchronize();
  
  
  // copying propag Vext to dev
  HANDLE_ERROR( hipMemcpyAsync(propagator_Vext_dev, propagator_Vext_host,
				NX*NY*NZ*sizeof(hipDoubleComplex),
				hipMemcpyHostToDevice,
				(streams)[HELPER_STREAM]) );
#ifdef V_DIP
  // copying propag Vext to dev
  HANDLE_ERROR( hipMemcpyAsync(Vdd_dev, Vdd_host,
				NX*NY*NZ*sizeof(hipDoubleComplex),
				hipMemcpyHostToDevice,
				(streams)[SIMULATION_STREAM]) );
#endif
  
#ifdef DEBUG
  /*
  HANDLE_ERROR( hipMemcpyAsync(propagator_T_host, propagator_T_dev,
				NX*NY*NZ*sizeof(hipDoubleComplex),
				hipMemcpyDeviceToHost,
				(streams)[HELPER_STREAM]) );
				*/
#endif
  
  // saving to file initial wavefuntion (1st frame) <- CZY TO JEST POTRZEBNE ???
  fwrite( wf_host, sizeof(double complex), NX*NY*NZ, (files[WF_FRAMES_FILE])->data );
  
  // saving to file propagators T, Vext, and F{ Vdd }
  /*
   * TODO: place this in helper thread on host
   */
  fprintf( (files[PROPAGATORS_FILE])->data, "x\t\t\tRe[e^-iVext(x)dt]\tIm[e^-iVext(x)dt]\tkx\t\t\tRe[e^-iT(kx)dt]\tIm[e^-iT(kx)dt]\tVdd\n"); // header
  for (uint64_t ii=0 ; ii < NX*NY*NZ; ii++) {
         fprintf( (files[PROPAGATORS_FILE])->data, "%.15f\t%.15f\t%.15f\t%.15f\t%.15f\t%.15f\t%.15f\n", XMIN + ii*DX, creal(propagator_Vext_host[ii]), cimag(propagator_Vext_host[ii]),
                                                                            kx(ii), creal(propagator_T_host[ii]), cimag(propagator_T_host[ii]),
                                                                            Vdd_host[ii]);
  }
//   for (uint64_t ii=0 ; ii < NX*NY*NZ/2; ii++)
//          fprintf( (files[PROPAGATORS_FILE])->data, "%.15f\t%.15f\t%.15f\n", ii, creal(propagator_Vext_host[ii]), cimag(propagator_Vext_host[ii]) );
  
#ifdef DEBUG
  
  printf("2nd barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
  printf("FLAG_RUN_SIMULATION %u\n",FLAG_RUN_SIMULATION);
#endif
  hipStreamSynchronize( (streams)[HELPER_STREAM] );
  pthread_barrier_wait (&barrier);
  
  
  
#ifdef DEBUG
  // checking total norm of propagator T (should be sqrt[Nx*|e^ia|^2] = sqrt[Nx], because Im[a] == 0 )
  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, propagator_T_dev, 1, norm_dev) );
  hipDeviceSynchronize();
  HANDLE_ERROR( hipMemcpyAsync(&norm_host, norm_dev,
				sizeof(double),
				hipMemcpyDeviceToHost,
				(streams)[HELPER_STREAM]) );
  hipDeviceSynchronize();
  fprintf( (files[PROPAGATORS_FILE])->data, "norm (cublas) propagator_T_dev: %.15f\n", norm_host );
#endif
  
  
  // checking norm of initial wavefunction
  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, norm_dev) );
  hipDeviceSynchronize();
  HANDLE_ERROR( hipMemcpyAsync(&norm_host, norm_dev,
        			sizeof(double),
				hipMemcpyDeviceToHost,
				(streams)[HELPER_STREAM]) );
  hipStreamSynchronize( streams[HELPER_STREAM] );
  norm_host *= sqrt(DX);  
  fprintf( (files[STATS_FILE])->data, "norm of initial wf: %.15f\tdx: %.15f\tsqrt dx: %.15f\n\n", norm_host, DX, sqrt(DX) );
  
  // header of a file with statistics
  fprintf( (files[STATS_FILE])->data, "\n\nt [dt]:\tnorm:\t\t\tchemic. pot.\t\t<T>\t\t\t<Vext>\t\t\t<Vcon>\t\t\t<Vdip>\n" );
  
  
       /* *************************************************************************************************************************************** *
	* 																	  *
	* 							ALGORITHM LOOP									  *
	* 																	  *
	* *************************************************************************************************************************************** */
  uint64_t saving_steps = frames_to_be_saved;
  uint64_t timesteps_tot = time_tot/frames_to_be_saved;
  uint64_t timesteps;
  
  printf("\n\n");
  printf("dt: %e\n",DT);
  printf("time total (in dt): %lu\n", time_tot);
  printf("time between saving (in dt): %lu\n", timesteps_tot);
  printf("savings: %lu\n", frames_to_be_saved);
  printf("\n");
  
  uint32_t counter = 0;
  while( FLAG_RUN_SIMULATION ) { // simulation will be runing until the flag is set to false
#ifdef DEBUG
     timesteps = 2;
     saving_steps =1;
     printf("timesteps to be made: %lu\n", timesteps);
#else
     timesteps = timesteps_tot;
     printf("%lu. timesteps to be made: %lu\n", saving_steps, timesteps);
#endif
     saving_steps--;
     
     while(timesteps) {
       timesteps--;
       /* *************************************************************************************************************************************** *
	* 																	  *
	* 							ALGORITHM STEP									  *
	* 																	  *
	* *************************************************************************************************************************************** */
       // multiply by Vext propagator (do in callback load) !*
       
       // make copy of wavefunction
       CHECK_CUBLAS( hipblasZcopy(cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, complex_arr3_dev, 1) );
       hipDeviceSynchronize();
       
       
       /*
        *       EVOLVE IN MOMENTUM SPACE
        */
       
       //printf("\ntransforming wavefunction to momentum space\n");
       CHECK_CUFFT( hipfftExecZ2Z((cufft_plans)[FORWARD_PSI],
				 complex_arr1_dev,
				 complex_arr2_dev,
				 HIPFFT_FORWARD) );
       
       // count |\psi|^2 array in meanwhile
       //hipStreamSynchronize(streams[SIMULATION_STREAM]);
       call_kernel_ZD_1d( ker_modulus_pow2_wf_1d, complex_arr3_dev, double_arr1_dev, streams[HELPER_STREAM]);
       // it could be replaced with complex_arr3_dev <- maybe faster to copy array with cublas and do not synchronize streams 
       
       // multiply by T propagator (do in callback) <- J
       call_kernel_ZZ_1d( ker_multiplyZZ, complex_arr2_dev, propagator_T_dev, (streams)[SIMULATION_STREAM] );
       
       
       
       // go back to 'positions`'? space <- JAK JEST PO ANGIELSKU PRZESTRZEN POLOZEN ???
       CHECK_CUFFT( hipfftExecZ2Z((cufft_plans)[BACKWARD_PSI],
				 complex_arr2_dev,
				 complex_arr1_dev,
				 HIPFFT_BACKWARD) );
       // run kernel to normalize aftter FFT
       call_kernel_Z_1d( ker_normalize_1d, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
       
       
#ifdef V_DIP
       hipStreamSynchronize(streams[HELPER_STREAM]); // ensure that density is counted 
       // count DFT of modulus of wavefunction (in positions` space).
       // CUFFT is done in HELPER_STREAM
       CHECK_CUFFT( hipfftExecD2Z((cufft_plans)[FORWARD_DIPOLAR],
				 double_arr1_dev,
				 complex_arr3_dev) ); // double to complex must be forward, so no need to specify direction
       // now in double_arr1_dev we have density in positions space represantation
       // and in complex_arr3_dev we have Fourier transform of density
#endif       
       
       /*
        *       EVOLVE IN POSITIONS` SPACE
        */
       
       // evolve via external potential Vext (if defined)
#ifdef V_EXT
       call_kernel_ZZ_1d( ker_multiplyZZ, complex_arr1_dev, propagator_Vext_dev, (streams)[SIMULATION_STREAM] );
#endif
       
       // evolve via contact interactions potential
#ifdef V_CON
 #ifndef VDIP
       hipStreamSynchronize(streams[HELPER_STREAM]); // make sure double_arr1_dev is filled with |\psi|^2
       call_kernel_ZD_1d( ker_propagate_Vcon_1d, complex_arr1_dev, double_arr1_dev,(streams)[SIMULATION_STREAM] );
 #endif
#endif
       
#ifdef VDIP
       // multiply Fourier Transform of density with Fourier transform of density
       call_kernel_ZD_1d( ker_multiplyZD, complex_arr3_dev, Vdd_dev, (streams)[HELPER_STREAM] ); // <- TODO: Do it in callback load !!!
       
       // count integral in potential of dipolar interactions - convolution (on complex_arr3_dev in place) 
       CHECK_CUFFT( hipfftExecZ2Z((cufft_plans)[BACKWARD_DIPOLAR],
				 complex_arr3_dev,
				 complex_arr3_dev,
				 HIPFFT_BACKWARD) );
       // normalize (not in callback store)
       
       // now in complex_arr3_dev we have Vint(r)
       
       // create propagator of Vdip (in) / propagate Vdip
       call_kernel_ZZ_1d( ker_propagate_Vint_1d, complex_arr1_dev, complex_arr3_dev, (streams)[SIMULATION_STREAM] );
#endif
       
       
       
#ifdef IMAG_TIME
       hipStreamSynchronize(streams[SIMULATION_STREAM]); // make sure evolution via interactions is completed
       // normalize wavefunction to |\psi|^2 = 1 (at every step!)
       CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, norm_dev) ); // count norm
       hipDeviceSynchronize(); // ensure norm is from current step
       call_kernel_ZD_1d( ker_normalize_1d, complex_arr1_dev, norm_dev, (streams)[SIMULATION_STREAM] ); // normalize
#endif
       
       /* *************************************************************************************************************************************** *
	* 																	  *
	* 							END OF ALGORITHM STEP								  *
	* 																	  *
	* *************************************************************************************************************************************** */
       
       
       
       // compute and save statistics of a system (norm, energy, ... )
#ifndef DEBUG
#ifdef REAL_TIME
       if ( counter%500 == 0 ) {
#endif
#ifdef IMAG_TIME
       if ( counter%50 == 0 ) {
#endif
#endif
         save_stats(counter);
         
#ifndef DEBUG
       }
#endif
       counter++;
       
     }     
       // saving wavefunction
       HANDLE_ERROR( hipMemcpy(wf_host, complex_arr1_dev, NX*NY*NZ*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
       /*for (uint64_t ii=0 ; ii < NX*NY*NZ; ii++)
         fprintf( ((files[])->data), "%.15f\t%.15f\t%.15f\n", XMIN+DX*ii, creal((wf_host)[ii]), cimag((wf_host)[ii]) );*/
       fwrite( wf_host, sizeof(double complex), NX*NY*NZ, (files[WF_FRAMES_FILE])->data );
       
       //save_stats(timesteps_tot-timesteps)*(frames_to_be_saved-saving_steps)
#ifdef DEBUG     
       //call_kernel_Z_1d( ker_print_Z, complex_arr1_dev, (streams)[SIMULATION_STREAM] );
#endif
     if (!saving_steps) FLAG_RUN_SIMULATION = false;
  }
  
  // saving wavefunction to binary file
  HANDLE_ERROR( hipMemcpy(wf_mmap, complex_arr1_dev, NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) ); // change to asynchronous!
  
#ifdef DEBUG
  printf("last barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
#endif
  hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
  
  // free memory on host
  HANDLE_ERROR( hipHostFree(wf_host) );
#ifdef DEBUG
  HANDLE_ERROR( hipHostFree(propagator_T_host) );
#endif
  
  free_device();
  /*
  HANDLE_ERROR( hipFree(complex_arr1_dev) ); 	//
  HANDLE_ERROR( hipFree(complex_arr2_dev) ); 	//
  //HANDLE_ERROR( hipFree(double_arr1_dev)  ); 	//
  HANDLE_ERROR( hipFree(propagator_T_dev) ); 	//
  //HANDLE_ERROR( hipFree(propagator_Vext_dev) );	//
  //HANDLE_ERROR( hipFree(Vdd_dev) );		//
  
  
  //HANDLE_ERROR( hipFree(global_stuff->mean_T_dev) ); // result of integral with kinetic energy operator in momentum representaion
  //HANDLE_ERROR( hipFree(mean_Vdip_dev) ); // result of integral with Vdip operator in positions' representation
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vext_dev) ); // result of integral with Vext operator in positions' representation
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vcon_dev) ); // result of integral with Vcon operator in positions' representation
  HANDLE_ERROR( hipFree(norm_dev) ); //
  */
  pthread_barrier_wait (&barrier_global);
  pthread_exit(NULL);
}





/* ************************************************************************************************************************************* *
 * 																	 *
 * 							HELPER THREAD									 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * - allocation memory on device
 * - allocation plans
 * 
 */
void* helper_thread(void* passing_ptr) {
  
  //stick_this_thread_to_core(2);
  
  pthread_barrier_wait (&barrier_global);
  printf("running %s thread.\n",thread_names[HELPER_THRD]);
  
  // init memory on device
  // arrays for wavefunction
  HANDLE_ERROR( hipMalloc((void**) &(complex_arr1_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	//
  HANDLE_ERROR( hipMalloc((void**) &(complex_arr2_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	//
  HANDLE_ERROR( hipMalloc((void**) &complex_arr3_dev, sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	//
  HANDLE_ERROR( hipMalloc((void**) &(double_arr1_dev), sizeof(double) * NX*NY*NZ) );		//
  
  // constant arrays
  HANDLE_ERROR( hipMalloc((void**) &(propagator_T_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	// array of constant factors e^-i*k**2/2*dt
  HANDLE_ERROR( hipMalloc((void**) &(propagator_Vext_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) );// array of constant factors e^-i*Vext*dt
  HANDLE_ERROR( hipMalloc((void**) &(Vdd_dev), sizeof(double) * NX*NY*NZ) ); 			// array of costant factors <- count on host with spec funcs lib or use Abramowitz & Stegun approximation
  
  // scalar variables
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->mean_T_dev), sizeof(double))    ); // result of integral with kinetic energy operator in momentum representaion
  HANDLE_ERROR( hipMalloc((void**) &meanZ_T_dev, sizeof(hipDoubleComplex))    ); // result of integral with kinetic energy operator in momentum representaion
  HANDLE_ERROR( hipMalloc((void**) &meanZ_Vext_dev, sizeof(hipDoubleComplex))    ); // result of integral with kinetic energy operator in momentum representaion
  HANDLE_ERROR( hipMalloc((void**) &meanZ_Vcon_dev, sizeof(hipDoubleComplex))    ); // result of integral with kinetic energy operator in momentum representaion
  HANDLE_ERROR( hipMalloc((void**) &(meanZ_Vdip_dev), sizeof(hipDoubleComplex)) ); // result of integral with Vdip operator in positions' representation
  HANDLE_ERROR( hipMalloc((void**) &(norm_dev), sizeof(double)) ); // variable to hold norm of wavefunction
  
  
#ifdef DEBUG
  printf("allocated memory on device.\n");
#endif
  
  for (uint8_t ii = 0; ii < num_streams; ii++)
    HANDLE_ERROR(	hipStreamCreate( &(streams[ii]) )	);
  
#ifdef DEBUG
  printf("1st barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  //hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
  
  // creating plans with callbacks
  cufft_plans = (hipfftHandle*) malloc( (size_t) sizeof(hipfftHandle)*num_plans );
#ifdef DEBUG
  printf("array of plans allocated.\n");
#endif
  for (uint8_t ii = 0; ii < num_plans; ii++) {
    CHECK_CUFFT(  hipfftCreate( (cufft_plans)+ii )  ); // allocates expandable plans
    //printf("%d\n",(cufft_plans)[ii]);
  }
  
#ifdef DEBUG
  printf("expandable plans allocated.\n");
#endif
  
  size_t work_size; // CHYBA TO MUSI BYC TABLICA !!!
#if (DIM == 1)
  printf("creating CUFFT plans in 1d case.\n");
  // wavefunction forward
  CHECK_CUFFT( hipfftMakePlan1d( (cufft_plans)[FORWARD_PSI], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size ) 	);
  CHECK_CUFFT( hipfftSetStream(  (cufft_plans)[FORWARD_PSI], (streams)[SIMULATION_STREAM] ) );
  //printf("%d\n",(cufft_plans)[FORWARD_PSI]);
  
  // wavefunction inverse
  //  printf("%p\n",(cufft_plans)+BACKWARD_PSI);
  CHECK_CUFFT( hipfftMakePlan1d( (cufft_plans)[BACKWARD_PSI], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (cufft_plans)[BACKWARD_PSI], (streams)[SIMULATION_STREAM]) );
  //printf("%d\n",(cufft_plans)[BACKWARD_PSI]);
  
  // modulus of wavefunction forward
  CHECK_CUFFT( hipfftMakePlan1d( (cufft_plans)[FORWARD_DIPOLAR], NX*NY*NZ, HIPFFT_D2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (cufft_plans)[FORWARD_DIPOLAR], (streams)[HELPER_STREAM] ) );
  //printf("%d\n",(cufft_plans)[FORWARD_DIPOLAR]);
  
  // integral in potential of dipolar inteaction
  CHECK_CUFFT( hipfftMakePlan1d( (cufft_plans)[BACKWARD_DIPOLAR], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (cufft_plans)[BACKWARD_DIPOLAR], (streams)[HELPER_STREAM]) ); // WLASCIWIE TUTAJ NIE WIADOMO W KTORYM STREAMIE?
  //printf("%d\n",(cufft_plans)[BACKWARD_DIPOLAR]);
  
#elif (DIM == 2)
  
#elif (DIM == 3)
  
#endif // case DIM for plan
  printf("\tplans created\n");
  
  // !!! SPRAWDZIC !!! funckje: <- co robia?
  //hipfftResult hipfftSetAutoAllocation(hipfftHandle *plan, bool autoAllocate);
  //cufftSetCompatibilityMode() <- musi byc wywolana po create a przed make plan
  
  
  /* ************************************
   * 			CUBLAS		*
   * ************************************/
  
  CHECK_CUBLAS( hipblasCreate(&cublas_handle) );
  CHECK_CUBLAS( hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) ); // this means we can use only device pointers to scalars (required by CUBLAS routines)
  
  printf("CUBLAS initialized!\n");
  
  
  
  
  
  
#ifdef DEBUG
  printf("created FFT plans.\n");
#endif
#ifdef DEBUG
  printf("2nd barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  pthread_barrier_wait (&barrier);
  
  save_simulation_params();
  
  // start algorithm
  // dt =
  /*uint16_t timesteps;
  while( FLAG_RUN_SIMULATION ) { // simulation will be runing until the flag is set to false
#ifdef DEBUG
     timesteps = 1;
#else
     timesteps = 1000;
#endif
     while(timesteps) {
       timesteps--;
       
       //FLAG_RUN_SIMULATION = false;
     }
  }*/
  
  //hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
#ifdef DEBUG
  printf("last barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  
  CHECK_CUBLAS( hipblasDestroy(cublas_handle) );
  //CHECK_CUBLAS( cublasShutdown() );
  
  
  
  
  for (uint8_t ii = 0; ii < num_streams; ii++)
    HANDLE_ERROR(	hipStreamDestroy( (streams[ii]) )	);
  
  pthread_barrier_wait (&barrier_global);
  pthread_exit(NULL);
}


void alloc_device(){
  
  
}


void alloc_host() {
  
  // must use
  
}

void free_device() {
  // free memory on device
  HANDLE_ERROR( hipFree(complex_arr1_dev) ); 	//
  HANDLE_ERROR( hipFree(complex_arr2_dev) ); 	//
  HANDLE_ERROR( hipFree(complex_arr3_dev) ); 	//
  HANDLE_ERROR( hipFree(double_arr1_dev)  ); 	//
  HANDLE_ERROR( hipFree(propagator_T_dev) ); 	//
  HANDLE_ERROR( hipFree(propagator_Vext_dev) );	//
  HANDLE_ERROR( hipFree(Vdd_dev) );		//
  
  
  //HANDLE_ERROR( hipFree(mean_T_dev) ); // result of integral with kinetic energy operator in momentum representaion
  //HANDLE_ERROR( hipFree(mean_Vdip_dev) ); // result of integral with Vdip operator in positions' representation
  //HANDLE_ERROR( hipFree(mean_Vext_dev) ); // result of integral with Vext operator in positions' representation
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vcon_dev) ); // result of integral with Vcon operator in positions' representation
  HANDLE_ERROR( hipFree(norm_dev) ); //
  
}


void free_pinnable() {
    
}

/* ************************************************************************************************************************************* *
 * 																	 *
 * 							SAVING FUNCTIONS								 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * Saving statistics of the system, assuming that complex_arr1 is the wavefunction after step_index-th iteration.
 * uint64_t step_index - index of iteration (timestep)
 */
void save_stats(uint64_t step_index) {
  // saving stats in pipelining mode
  
  // count norm with cublas (it is already done in case of imaginary-time evolution)
#ifndef IMAG_TIME
  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, norm_dev) );
#endif
  
  hipDeviceSynchronize();
  
  // count <T> and copy norm in parallel
  CHECK_CUFFT( hipfftExecZ2Z((cufft_plans)[FORWARD_PSI], complex_arr1_dev, complex_arr2_dev, HIPFFT_FORWARD) ); // make sure we have copy of wavefunction in momentum space
  HANDLE_ERROR( hipMemcpyAsync(&norm_host, norm_dev, sizeof(double), hipMemcpyDeviceToHost, (streams)[HELPER_STREAM]) ); // copy norm to host in parallel
  hipStreamSynchronize( (streams)[SIMULATION_STREAM] ); // make sure FFT done
  call_kernel_ZZ_1d( ker_T_wf, complex_arr2_dev, complex_arr3_dev, (streams)[SIMULATION_STREAM]); // make vector T|\psi>
  hipStreamSynchronize( (streams)[SIMULATION_STREAM] ); // make sure FFT done
  CHECK_CUBLAS( hipblasZdotc(cublas_handle, NX*NY*NZ, (complex_arr2_dev), 1, complex_arr3_dev, 1, meanZ_T_dev) ); // count <\psi||T \psi>
  hipDeviceSynchronize();
  
  // count <Vext> and copy <T> in parallel
#ifdef V_EXT
  call_kernel_ZZ_1d( ker_Vext_wf, complex_arr1_dev, complex_arr3_dev, (streams)[SIMULATION_STREAM]); // make vector Vext|\psi>
#endif
  HANDLE_ERROR( hipMemcpyAsync(&meanZ_T_host, meanZ_T_dev, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, (streams)[HELPER_STREAM]) ); // copy <T> in parallel
  hipStreamSynchronize( (streams)[SIMULATION_STREAM] ); // make sure |Vext \psi> is done
#ifdef V_EXT
  CHECK_CUBLAS( hipblasZdotc(cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, complex_arr3_dev, 1, meanZ_Vext_dev) ); // count <\psi||T \psi>
#endif
  hipDeviceSynchronize();
  
  // count <Vcon> and copy <Vext> in parallel
#ifdef V_CON
  call_kernel_ZDZ_1d( ker_Vcon_wf, complex_arr1_dev, double_arr1_dev, complex_arr3_dev, streams[SIMULATION_STREAM] );
#endif
#ifdef V_EXT
  HANDLE_ERROR( hipMemcpyAsync(&meanZ_Vext_host, meanZ_Vext_dev, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, (streams)[HELPER_STREAM]) ); // copy <Vext> in parallel
#endif
  hipStreamSynchronize( (streams)[SIMULATION_STREAM] );
#ifdef V_CON
  CHECK_CUBLAS( hipblasZdotc(cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, complex_arr3_dev, 1, meanZ_Vcon_dev) ); // count <\psi||T \psi>
#endif
  hipDeviceSynchronize();
  
  // count <Vdip> and <Vcon> in parallel
#ifdef V_DIP
  
#endif
#ifdef V_CON
  HANDLE_ERROR( hipMemcpyAsync(&meanZ_Vcon_host, meanZ_Vcon_dev, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, (streams)[HELPER_STREAM]) ); // copy <Vcon> in parallel
#endif
  hipStreamSynchronize( (streams)[SIMULATION_STREAM] );
#ifdef V_DIP
  //CHECK_CUBLAS( hipblasZdotc(cublas_handle, NX*NY*NZ, complex_arr1_dev, 1, complex_arr3_dev, 1, meanZ_Vdip_dev) ); // count <\psi||T \psi>
#endif  
  
  
  
  
  //hipDeviceSynchronize(); <- not necessary
  norm_host *= sqrt(DX);
#ifdef IMAG_TIME
  chemical_potential_host = chemical_potential_ite(norm_host);
#else
  chemical_potential_host = 0.;
#endif
  //mean_T_host = creal(meanZ_T_host);
  meanZ_T_host *= (0.5*DX/NX) + I*0.;
#ifdef V_EXT
  meanZ_Vext_host *= (0.5*DX*OMEGA*OMEGA) + I*0.;
#else
  meanZ_Vext_host = 0. + I*0.;
#endif
#ifdef V_CON
  meanZ_Vcon_host *= (0.5*DX);
#else
  meanZ_Vcon_host = 0. + I*0.;
#endif
#ifdef V_DIP
  
#else
  meanZ_Vdip_host = 0. + I*0.;
#endif
  
  double Energy_tot = creal(meanZ_T_host) + creal(meanZ_Vext_host);
#ifdef DEBUG
  printf("T:\t%.15f + %.15fj\n",creal(meanZ_T_host), cimag(meanZ_T_host));
  printf("Vext:\t%.15f + %.15fj\n",creal(meanZ_Vext_host), cimag(meanZ_Vext_host));
  printf("Etot:\t%.15f\n",Energy_tot);
#endif
  
  fprintf( ((files[STATS_FILE])->data), "%lu.\t%.15f\t%.15f\t%.15f\t%.15f\t%.15f\t%.15f\n", step_index,
								    norm_host,
                                                                    chemical_potential_host,
								    creal(meanZ_T_host),
								    creal(meanZ_Vext_host),
                                                                    creal(meanZ_Vcon_host),
                                                                    creal(mean_Vdip_host));
}


/*
 * Saves simulations parameters to special file.
 * 
 * !!! CURRENTLY INVOKED IN HELPER THREAD AFTER 2ND BARRIER !!!
 */
void save_simulation_params() {
    char str_date[17];
    time_t t = time(NULL);
    strftime(str_date, sizeof(str_date), "%Y-%m-%d_%H:%M", localtime(&t)); 
    
    //fprintf( (files[SIM_PARAMS_FILE])->data,"\t\t\tSIMULATION GPE\n");
    fprintf( (files[SIM_PARAMS_FILE])->data,"date:\t%s\n",str_date);
    uint8_t dim = DIM;
    fprintf( (files[SIM_PARAMS_FILE])->data,"dim:\t%u\n", dim );
#ifdef IMAG_TIME
    fprintf( (files[SIM_PARAMS_FILE])->data, "evolution:\t%s\n","imaginary time" );
#else
    fprintf( (files[SIM_PARAMS_FILE])->data, "evolution:\t%s\n","real time" );
#endif
#ifdef V_EXT
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vext:\t%s\n","yes" );
#else
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vext:\t%s\n","no" );
#endif
#ifdef V_CON
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vcon:\t%s\n","yes" );
#else
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vcon:\t%s\n","no" );
#endif
#ifdef V_DIP
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vdip:\t%s\n","yes" );
#else
    fprintf( (files[SIM_PARAMS_FILE])->data, "Vdip:\t%s\n","no" );
#endif
    fprintf( (files[SIM_PARAMS_FILE])->data,"timesteps made:\t%u\n", time_tot );
    fprintf( (files[SIM_PARAMS_FILE])->data,"dt:\t%.15f\n", DT );
    fprintf( (files[SIM_PARAMS_FILE])->data,"frames:\t%u\n", frames_to_be_saved );
    fprintf( (files[SIM_PARAMS_FILE])->data,"Nx:\t%u\n", NX );
    fprintf( (files[SIM_PARAMS_FILE])->data,"Ny:\t%u\n", NY );
    fprintf( (files[SIM_PARAMS_FILE])->data,"Nz:\t%u\n", NZ );
    fprintf( (files[SIM_PARAMS_FILE])->data,"N:\t%u\n", NX*NY*NZ );
    fprintf( (files[SIM_PARAMS_FILE])->data,"xmin:\t%.15f\n", XMIN );
    fprintf( (files[SIM_PARAMS_FILE])->data,"xmax:\t%.15f\n", XMAX );
    fprintf( (files[SIM_PARAMS_FILE])->data,"dx:\t%.15f\n",DX );
    fprintf( (files[SIM_PARAMS_FILE])->data,"ymin:\t%.15f\n", 0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"ymax:\t%.15f\n", 0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"dy: %.15f\n",0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"zmin:\t%.15f\n", 0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"zmax:\t%.15f\n", 0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"dz: %.15f\n",0. );
    fprintf( (files[SIM_PARAMS_FILE])->data,"kx_min:\t%.15f\n", KxMIN );
    fprintf( (files[SIM_PARAMS_FILE])->data,"kx_max:\t%.15f\n", KxMAX );
    fprintf( (files[SIM_PARAMS_FILE])->data,"dkx:\t%.15f\n",DKx );
    //printf("width of gauss in positions space (points on lattice): %.15f\n");
    //printf("width of gauss in positions space (points on lattice): %.15f\n");
#ifdef OMEGA
    fprintf( (files[SIM_PARAMS_FILE])->data,"harmonic potential angular freq.:\t%.15f\n", OMEGA );
#endif
#ifdef G_CONTACT
    fprintf( (files[SIM_PARAMS_FILE])->data,"contact interactions g factor:\t%.15f\n", G_CONTACT );
#endif
#ifdef G_DIPOLAR
    fprintf( (files[SIM_PARAMS_FILE])->data,"dipolar interactions g factor:\t%.15f\n", G_DIPOLAR );
#endif
    
    // here some more ...
}