#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#define N ((uint64_t) 1<<22)

#define M_2PI (6.283185307179586)
#define SQRT_2PI (2.5066282746310002)
#define INV_SQRT_2PI (0.3989422804014327)
#define SIGMA (1)

/*
 * compile: 	nvcc -dc -lm -g -G -Xptxas="-v" -m64 -O3 -o simple_example.o -c simple_example.cu
 * 		nvcc -m64 -arch=sm_35 -o simple_example simple_example.o -lcufft_static -lculibos
 * 
 *		//with store callback to normalize: 
 * 		nvcc -dc -lm -g -G -Xptxas="-v" -m64 -O3 -o simple_example_cb_store_normalize.o -c simple_example.cu -D STORE_CB_NORMALIZE
 * 		nvcc -m64 -arch=sm_35 -o simple_example_cb_store_normalize simple_example_cb_store_normalize.o -lcufft_static -lculibos
 * 
 * sources:
 * http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/
 * + example in cuda samples
 */

/* CALLBACK TYPES

typedef enum hipfftXtCallbackType_t {
    HIPFFT_CB_LD_COMPLEX = 0x0,
    HIPFFT_CB_LD_COMPLEX_DOUBLE = 0x1,
    HIPFFT_CB_LD_REAL = 0x2,
    HIPFFT_CB_LD_REAL_DOUBLE = 0x3,
    HIPFFT_CB_ST_COMPLEX = 0x4,
    HIPFFT_CB_ST_COMPLEX_DOUBLE = 0x5,
    HIPFFT_CB_ST_REAL = 0x6,
    HIPFFT_CB_ST_REAL_DOUBLE = 0x7,
    HIPFFT_CB_UNDEFINED = 0x8
} hipfftXtCallbackType;


Read more at: http://docs.nvidia.com/cuda/cufft/index.html#ixzz3fPgy3600
Follow us: @GPUComputing on Twitter | NVIDIA on Facebook

*/


static __device__ hipfftDoubleComplex cufftSgn(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
    if (offset < N/2) {
      //((hipfftDoubleComplex* ) dataIn)[offset] = make_hipDoubleComplex(-1.,0.);
      //printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(-1.,0.);
    }
    else {
      
      //printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(1.,0.);
    }
}

static __device__ hipfftDoubleComplex cufftRect(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
    if ((offset < N/4) || offset > 3*N/4) {
      //((hipfftDoubleComplex* ) dataIn)[offset] = make_hipDoubleComplex(-1.,0.);
      //printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(0.,0.);
    }
    else {
      
      //printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(1.,0.);
    }
}

static __device__ hipfftDoubleComplex cudaGauss_1d(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
  // get the index of thread
  //uint64_t ii = offset;
  
  
  // allocate constants in shared memory
  const double x0 = (-5*SIGMA);
  const double dx = (10*SIGMA)/((double) N);
  
  //if (ii < N) {
  //  ((hipfftDoubleComplex)* dataIn)[ii] = make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + ii*dx)*(x0 + ii*dx)/2/SIGMA)/SIGMA, 0. );
    
  //}
  return make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + offset*dx)*(x0 + offset*dx)/2/SIGMA)/SIGMA, 0. );
}

static __device__ void cufft_normalize(void *dataOut, 
				  size_t offset, 
				  hipfftDoubleComplex element, 
				  void *callerInfo, 
				  void *sharedPtr) 
{
  ((hipfftDoubleComplex*) dataOut)[offset] = make_hipDoubleComplex( hipCreal(element)/((double) N), hipCimag(element)/((double) N) );
}
// pointer to callback function (on device)
//__device__ hipfftCallbackLoadZ d_loadCallbackPtr = cudaGauss_1d;
__device__ hipfftCallbackLoadZ d_loadCallbackPtr = cufftRect;
__device__ hipfftCallbackStoreZ d_storeCallbackPtr = cufft_normalize;

__global__ void kernel_normalize(hipfftDoubleComplex* cufft_inverse_data) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // in both kernel as well as callback we use predefined N to have comparable performance results
  
  if (ii < N) {
    cufft_inverse_data[ii] = make_hipDoubleComplex( hipCreal(cufft_inverse_data[ii])/((double) N), hipCimag(cufft_inverse_data[ii])/((double) N) );
  }
}

int main (){
  
  hipDeviceReset();
  hipDeviceSynchronize();
  
  const uint8_t filename_str_lenght = 128;
  const uint8_t dim = 1;
    
  char filename1d[filename_str_lenght];
  FILE *file1d;
  
#ifdef STORE_CB_NORMALIZE
  sprintf(filename1d,"cufft_%dd_N%lu_cb_store.bin",dim,N );
#else
  sprintf(filename1d,"cufft_%dd_N%lu.bin",dim,N );
#endif
  printf("1d cufft example save in: %s\n",filename1d);
  file1d = fopen(filename1d, "wb");
  if (file1d == NULL)
  {
      printf("Error opening file %s!\n",filename1d);
      exit(EXIT_FAILURE);
  }
  
  printf("N %lu\n",N);
  
  hipfftDoubleComplex *data_dev;
  hipMalloc((void**)&data_dev, sizeof(hipfftDoubleComplex)*N);
  hipDeviceSynchronize();
  
  hipfftDoubleComplex* data_host;
  hipHostAlloc((void**) &data_host, sizeof(hipfftDoubleComplex)*N, hipHostMallocDefault);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    exit(EXIT_FAILURE);
  }
  hipDeviceSynchronize();
  
  
  // get host-usable pointer to callback functions
  hipfftCallbackLoadZ h_loadCallbackPtr;
  hipMemcpyFromSymbol(&h_loadCallbackPtr, 
                       HIP_SYMBOL(d_loadCallbackPtr), 
                       sizeof(h_loadCallbackPtr) );
  hipDeviceSynchronize();
  
  
  
  // creating plan with callback
  hipfftHandle plan;
  hipfftCreate(&plan);
  
  size_t work_size;
  hipfftMakePlan1d(plan, N, HIPFFT_Z2Z, 1, &work_size); //hipfftMakePlan1d(hipfftHandle *plan, int nx, hipfftType type, int batch)
  
  /*if (hipfftPlan1d(&plan, N, HIPFFT_C2C, 1) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(EXIT_FAILURE);
  }*/
  hipDeviceSynchronize();
  hipfftResult status = hipfftXtSetCallback(plan,
		     (void **) &h_loadCallbackPtr,
                     HIPFFT_CB_LD_COMPLEX_DOUBLE,
                     NULL ); //<- here can be added structure with data needed for callback execution!
  if (status == CUFFT_LICENSE_ERROR)
  {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
  }
  
  
  
  
  
  
  
  
  
  
  
  hipfftCallbackStoreZ h_storeCallbackPtr;
  hipMemcpyFromSymbol(&h_storeCallbackPtr, 
                       HIP_SYMBOL(d_storeCallbackPtr), 
                       sizeof(h_storeCallbackPtr));
  
  
  // creating plan with callback
  hipfftHandle plan_back;
  hipfftCreate(&plan_back);
  
  size_t work_size2;
  hipfftMakePlan1d(plan_back, N, HIPFFT_Z2Z, 1, &work_size2); //hipfftMakePlan1d(hipfftHandle *plan, int nx, hipfftType type, int batch)
  
  hipDeviceSynchronize();
#ifdef STORE_CB_NORMALIZE
  status = hipfftXtSetCallback(plan_back,
		     (void **) &h_storeCallbackPtr,
                     HIPFFT_CB_ST_COMPLEX_DOUBLE,
                     NULL ); //<- here can be added structure with data needed for callback execution!
  if (status == CUFFT_LICENSE_ERROR)
  {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
  }
#endif
  hipDeviceSynchronize();
  
  
  
  // EXECUTE TRANSFORM FORWARD AND BACKWADR
  
  if (hipfftExecZ2Z(plan, data_dev, data_dev, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
    exit(EXIT_FAILURE);;
  }
  hipDeviceSynchronize();
  
  
  // copy data
  hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("fft data forward:\n");
  for (uint64_t ii = 0; ii < N; ii++) {
    if (N <= 32) printf("%lf + %lfj\n", hipCreal(data_host[ii]), hipCimag(data_host[ii]));
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  
  
  // execute back
  if (hipfftExecZ2Z(plan_back, data_dev, data_dev, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
    exit(EXIT_FAILURE);;
  }
  hipDeviceSynchronize();
#ifndef STORE_CB_NORMALIZE
  // run kernel to normalize
  uint64_t threadsPerBlock;
  if (N >= 33554432)
    threadsPerBlock = 1024;
  else {
    threadsPerBlock = 128; // seems max grid size is ( 32768, ?, ? ) <- ????
  }
  dim3 dimBlock(threadsPerBlock,1,1);
  dim3 dimGrid( (N + threadsPerBlock - 1)/threadsPerBlock, 1, 1 ); // (numElements + threadsPerBlock - 1) / threadsPerBlock
  printf("initating wavefunction on host. Kernel invocation:\n");
  printf("threads Per block: %lu\n", threadsPerBlock);
  printf("blocks: %lu\n",(N + threadsPerBlock - 1)/threadsPerBlock);
  // filling with data
  kernel_normalize<<<dimGrid,dimBlock>>>(data_dev);
  //HANDLE_ERROR( hipGetLastError() );
#endif
  
  // copy data
  hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("fft data backward:\n");
  for (uint64_t ii = 0; ii < N; ii++) {
    if (N <= 32) printf("%lf + %lfj\n", hipCreal(data_host[ii]), hipCimag(data_host[ii]));
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  
  
  fclose(file1d);
  
  hipFree(data_dev);
  hipHostFree(data_host);
  hipDeviceSynchronize();
  
  hipDeviceReset();
  hipDeviceSynchronize();
  
  return EXIT_SUCCESS;
}