#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>



__global__ void staticReverse(hipDoubleComplex *d, int n)
{
  __shared__ hipDoubleComplex s[64];
  int t = threadIdx.x + blockIdx.x*blockDim.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(hipDoubleComplex *d, int n)
{
  extern __shared__ hipDoubleComplex s[];
  int t = threadIdx.x + blockIdx.x*blockDim.x;
  int tr = n-t-1;
  printf("tr: %d",tr);
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64*512; // 2**15
  hipDoubleComplex a[n], r[n], d[n];
  
  for (int i = 0; i < n; i++) {
    a[i] = make_hipDoubleComplex( (double) i, 0. );
    r[i] = make_hipDoubleComplex( (double) n-i-1, 0. );
    d[i] = make_hipDoubleComplex( 0. , 0.);
  }
  
  hipDoubleComplex *d_d;
  hipMalloc(&d_d, n * sizeof(hipDoubleComplex)); 
  
  printf("static reverse\n");
  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  staticReverse<<<512,n/512>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if ( hipCreal(d[i]) != hipCreal(r[i]) ) printf("Error: d[%d]!=r[%d] (%lf, %lf)n", i, i,hipCreal(d[i]), hipCreal(r[i])); 
  
  printf("dynamic reverse\n");
  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  
  // WYTESTOWAC odpalenie tego kernela z parametrami dimGrid(512,1,1)/dimBlocks(128,1,1)/3KB = 3*(1<<10) KB Shared Mem per thread !!!
  printf("sizeof cuDoubleComplex: %d",sizeof(hipDoubleComplex));
  printf("memory needed: %d",n*sizeof(hipDoubleComplex));
  dynamicReverse<<<512,n/512,n*sizeof(hipDoubleComplex)/512>>>(d_d, n); 
  
  
  hipMemcpy(d, d_d, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if ( hipCreal(d[i]) != hipCreal(r[i]) ) printf("Error: d[%d]!=r[%d] (%lf, %lf)n", i, i,hipCreal(d[i]), hipCreal(r[i]));
}
