#include  <hipfft/hipfft.h> 


#define  NX  256 
#define  BATCH  10 



void cuda_dft_1d() {
  
  hipfftHandle plan; 
  hipfftComplex *data; 
  
  hipMalloc((void**)&data,  sizeof(hipfftComplex)*NX*BATCH );
  
  /* Create   a  1D  FFT  plan. */ 
  hipfftPlan1d(&plan, NX, CUFFT C2C, BATCH);
  
  /* Use  the  CUFFT  plan  to  transform  the  signal  in place. */ 
  hipfftExecC2C(plan, data, data, CUFFT FORWARD); 
  
  /* Destroy  the  CUFFT  plan. */ 
  hipfftDestroy(plan); 
  hipFree(data);
}

/*
#define NX 64
#define NY 64
#define NZ 128

hipfftHandle plan;
hipfftComplex *data1, *data2;
hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY*NZ);
hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY*NZ);
// Create a 3D FFT plan. 
hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

// Transform the first signal in place.
hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

// Transform the second signal using the same plan.
hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

// Destroy the cuFFT plan.
hipfftDestroy(plan);
hipFree(data1); hipFree(data2);
*/