#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>
#include <hip/driver_types.h>

#include "cuda_dft.cuh"
#include "book.h"

#include "cudautils.cuh"

// timing
hipEvent_t start;
hipEvent_t stop;


__global__ void cudaGauss_1d(hipfftDoubleComplex* data, const unsigned long long N) {
  // get the index of thread
  unsigned long long ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // allocate constants in shared memory
  const double x0 = (-5*SIGMA);
  const double dx = (10*SIGMA)/((double) N);
  
  if (ii < N) {
    data[ii] = make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + ii*dx)*(x0 + ii*dx)/2/SIGMA)/SIGMA, 0. );
  }
  
  __syncthreads();
  //printf("Kernel sie wykonuje\n");
}


void perform_cufft_1d(const uint64_t N, FILE** array_timing) {
  
  // initilizing files to save data
  const uint8_t filename_str_lenght = 128;
  const uint8_t dim = 1;
    
  char filename1d[filename_str_lenght];
  FILE *file1d;
  
  sprintf(filename1d,"./data/cufft_%dd_N%lu.bin",dim,N );
  printf("1d cufft example save in: %s\n",filename1d);
  file1d = fopen(filename1d, "wb");
  if (file1d == NULL)
  {
      printf("Error opening file %s!\n",filename1d);
      exit(EXIT_FAILURE);
  }
  
  
  
  
  // allocate memory
  hipfftDoubleComplex *data_dev;
  hipfftDoubleComplex *data_host;
  printf("sizeof(hipfftDoubleComplex): %lu\n", sizeof(hipfftDoubleComplex));
  printf("memory: %lu kB\n", sizeof(hipfftDoubleComplex)*N/1024);
  HANDLE_ERROR( hipMalloc((void**) &data_dev, sizeof(hipfftDoubleComplex)*N) );
  if (N < 65536) {
    HANDLE_ERROR( hipHostAlloc((void**) &data_host, sizeof(hipfftDoubleComplex)*N, hipHostMallocDefault) ); // when to use pinned memory: http://www.cs.virginia.edu/~mwb7w/cuda_support/pinned_tradeoff.html
  }
  else {
    data_host = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*N);
  }
  
  
  // fill array
  int threadsPerBlock = 512;
  printf("%lu\n",(N + threadsPerBlock - 1)/threadsPerBlock);
  dim3 dimGrid( (N + threadsPerBlock - 1)/threadsPerBlock, 1, 1 ); // (numElements + threadsPerBlock - 1) / threadsPerBlock
  dim3 dimBlock(threadsPerBlock,1,1);
  cudaGauss_1d<<<dimGrid,dimBlock>>>(data_dev, N);
  HANDLE_ERROR( hipGetLastError() );
  
  
  HANDLE_ERROR( hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipDeviceSynchronize() );
  
  if (N < 65536) {
  for (uint16_t ii = 0; ii < N; ii++) {
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  }
  
  hipfftHandle plan_forward;
  CUDATIMEIT_START;
  hipfftPlan1d(&plan_forward, N, HIPFFT_Z2Z, 1); // N - samples in array, 1 - number of arrays, must be splitted
  //hipfftPlan1d(&plan_forward, (N <= 512) ? N : 512, HIPFFT_Z2Z, (N <= 512) ? 1 : N/512); Maybe there is more efficient way ???
  CUDATIMEIT_STOP;
  fprint_cudatimeit(array_timing[1]);
  
  // inplace
  CUDATIMEIT_START;
  if (hipfftExecZ2Z(plan_forward, data_dev, data_dev, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed!\n");
    exit( EXIT_FAILURE );
  }
  CUDATIMEIT_STOP;
  fprint_cudatimeit(array_timing[1]);
  
  
  HANDLE_ERROR( hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipDeviceSynchronize() );
  
  for (uint16_t ii = 0; ii < N; ii++) {
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  
  //  cleaning up the mesh
  HANDLE_ERROR( hipFree(data_dev) );
  HANDLE_ERROR( hipHostFree(data_host) );
  
}

/*
// 


#define NX 64
#define NY 64
#define NZ 128

hipfftHandle plan;
hipfftComplex *data1, *data2;
hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY*NZ);
hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY*NZ);
// Create a 3D FFT plan. 
hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

// Transform the first signal in place.
hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

// Transform the second signal using the same plan.
hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

// Destroy the cuFFT plan.
hipfftDestroy(plan);
hipFree(data1); hipFree(data2);
*/
