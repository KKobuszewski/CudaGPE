#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <pthread.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "global.h"
#include "simulation.cuh"
#include "cudautils.cuh"
#include "kernels.cuh"


// timing
hipEvent_t start_t;
hipEvent_t stop_t;


// global variables
extern Globals* global_stuff;
bool FLAG_RUN_SIMULATION = true;
extern const char* thread_names[];
extern const char* stream_names[];
//extern pthread_barrier_t barrier;


pthread_barrier_t barrier;
hipblasHandle_t cublas_handle;

/*
 * 
 * !!! VERSION FOR 1 PTHREAD !!!
 * 
 */


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							SIM THREAD									 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * - allocation memory on host
 * - initialization of data
 * - main algorithm
 */
void* simulation_thread(void* passing_ptr) {
  
  double complex* propagator_T_host;
  double complex* propagator_Vext_host;
  double norm_host;
  
  //stick_this_thread_to_core(1); <- in cudautils, not used, include to header first
  pthread_barrier_wait (&barrier_global);
  printf("running %s thread.\n",thread_names[SIMULATION_THRD]);
  
  // allocate memory on host
  hipHostAlloc((void**) &(global_stuff->wf_host), sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  hipHostAlloc((void**) &norm_host, sizeof(double), hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  hipHostAlloc((void**) &propagator_T_host, sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc
  hipHostAlloc((void**) &propagator_Vext_host, sizeof(double complex)*NX*NY*NZ, hipHostMallocDefault); // pinnable memory <- check here for hipHostMalloc (could be faster)
  printf("allocated memory on host.\n");
  
  
  // fill arrays on host & device
//   if (global_stuff->init_wf_fd != -1) {
//     for (uint64_t ii = 0; ii < NX*NY*NZ; ii++) {
//       global_stuff->wf_host[ii] = global_stuff->init_wf_map[ii];
//     }
//   }
  
  
#ifdef DEBUG
  printf("1st barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
#endif
  pthread_barrier_wait (&barrier);
  // copy data async from host to device (if needed)
  if (global_stuff->init_wf_fd != -1) {
    // copy data from host to device (if needed) / cannot async because
    printf("copying initial wavefunction on device");
    HANDLE_ERROR( hipMemcpy(global_stuff->complex_arr1_dev, global_stuff->init_wf_map, NX*NY*NZ * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice) ); // change to asynchronous!
  }
  else {
    
    printf("initating wavefunction on device.\n");
    call_kernel_Z_1d( ker_gauss_1d, global_stuff->complex_arr1_dev, (global_stuff->streams)[SIMULATION_STREAM] );
    
  }
  
  printf("creating propagator T\n");
  //call_kernel_Z_1d( ker_create_propagator_T, global_stuff->propagator_T_dev, (global_stuff->streams)[HELPER_STREAM] );
  for( uint64_t ii=0; ii < NX; ii++ ) {
    propagator_T_host[ii] = cexp(-I*kx(ii)*(0.5*kx(ii)*DT));
    propagator_Vext_host[ii] = cexp(-I*(0.5*OMEGA)*(OMEGA*(ii*DX+XMIN))*((ii*DX+XMIN)*DT)); // <- !!! KOLEJNOSC MNOZEMIA A DOKLADNOSC !!!
    //printf("%.15f + %.15fj\n",creal(propagator_Vext_host[ii]), cimag(propagator_Vext_host[ii]) );
  }
  
  // copying propag T to dev
  HANDLE_ERROR( hipMemcpyAsync(global_stuff->propagator_T_dev, propagator_T_host,
				NX*NY*NZ*sizeof(hipfftDoubleComplex),
				hipMemcpyHostToDevice,
				(global_stuff->streams)[HELPER_STREAM]) );
  
  // copying after initialization (in meantime on another stream)
  HANDLE_ERROR( hipMemcpyAsync(global_stuff->wf_host, global_stuff->complex_arr1_dev,
				NX*NY*NZ*sizeof(hipfftDoubleComplex),
				hipMemcpyDeviceToHost,
				(global_stuff->streams)[SIMULATION_STREAM]) );
  hipDeviceSynchronize();
  
  
  // copying propag Vext to dev
  HANDLE_ERROR( hipMemcpyAsync(global_stuff->propagator_Vext_dev, propagator_Vext_host,
				NX*NY*NZ*sizeof(hipfftDoubleComplex),
				hipMemcpyHostToDevice,
				(global_stuff->streams)[SIMULATION_STREAM]) );
  
#ifdef DEBUG
  HANDLE_ERROR( hipMemcpyAsync(propagator_T_host, global_stuff->propagator_T_dev,
				NX*NY*NZ*sizeof(hipfftDoubleComplex),
				hipMemcpyDeviceToHost,
				(global_stuff->streams)[HELPER_STREAM]) );
  
  
#endif
  
  // saving to file after initialization
  for (uint64_t ii=0 ; ii < NX*NY*NZ; ii++)
         fprintf( (global_stuff->files)[0], "%.15f\t%.15f\t%.15f\n", XMIN+DX*ii, creal((global_stuff->wf_host)[ii]), cimag((global_stuff->wf_host)[ii]) );
  for (uint64_t ii=NX*NY*NZ/2 ; ii < NX*NY*NZ; ii++)
         fprintf( (global_stuff->files)[4], "%.15f\t%.15f\t%.15f\n", kx(ii), creal(propagator_T_host[ii]), cimag(propagator_T_host[ii]) );
  for (uint64_t ii=0 ; ii < NX*NY*NZ/2; ii++)
         fprintf( (global_stuff->files)[4], "%.15f\t%.15f\t%.15f\n", kx(ii), creal(propagator_T_host[ii]), cimag(propagator_T_host[ii]) );
  
#ifdef DEBUG
  
  printf("2nd barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
  printf("FLAG_RUN_SIMULATION %u\n",FLAG_RUN_SIMULATION);
#endif
  hipStreamSynchronize( (global_stuff->streams)[HELPER_STREAM] );
  pthread_barrier_wait (&barrier);
  
  
#ifdef DEBUG     
  call_kernel_Z_1d( ker_print_Z, global_stuff->complex_arr1_dev, (global_stuff->streams)[SIMULATION_STREAM] );
#endif
  // checking norm of initial wavefunction
  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, global_stuff->complex_arr1_dev, 1, global_stuff->norm_dev) );
  hipDeviceSynchronize();
  
  HANDLE_ERROR( hipMemcpyAsync(&norm_host, global_stuff->norm_dev,
        			sizeof(double),
				hipMemcpyDeviceToHost,
				(global_stuff->streams)[HELPER_STREAM]) );
  hipDeviceSynchronize();
  norm_host *= sqrt(DX);
  fprintf( (global_stuff->files)[3], "norm of initial wf: %.15f\tdx: %.15f\tsqrt dx: %.15f\n\n", norm_host, DX, sqrt(DX) );
  
  // checking total norm of propagator T (should be sqrt[Nx*|e^ia|^2] = sqrt[Nx], because Im[a] == 0 )
  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, global_stuff->propagator_T_dev, 1, global_stuff->norm_dev) );
  hipDeviceSynchronize();
  HANDLE_ERROR( hipMemcpyAsync(&norm_host, global_stuff->norm_dev,
				sizeof(double),
				hipMemcpyDeviceToHost,
				(global_stuff->streams)[HELPER_STREAM]) );
  hipDeviceSynchronize();
  fprintf( (global_stuff->files)[3], "norm (cublas) propagator_T_dev: %.15f\n", norm_host );
       
  // header of a file <- DO FILEIO.C PRZENIESC!
  fprintf( (global_stuff->files)[3], "\ntimestep:\tnorm after (cublas):\n" );
  
  
  // start algorithm
  // dt =
  //const uint64_t time_tot = llround(0.318309886183791/DT); // no Vext revival time
  const uint64_t time_tot = 10*llround((4*3.14159265358979323846/OMEGA)/DT); // harmonic potential revival time
  const uint64_t saving_tot = 20;
  uint64_t saving_steps = saving_tot;
  uint64_t timesteps_tot = time_tot/saving_tot;
  uint64_t timesteps;
  
  printf("\n\n");
  printf("time total (in dt): %lu\n", time_tot);
  printf("time between saving (in dt): %lu\n", timesteps_tot);
  printf("savings: %lu\n", saving_tot);
  printf("\n");
  
  while( FLAG_RUN_SIMULATION ) { // simulation will be runing until the flag is set to false
#ifdef DEBUG
     timesteps = 1;
     printf("timesteps to be made: %u\n", timesteps);
#else
     timesteps = timesteps_tot;
     printf("%lu. timesteps to be made: %lu\n", saving_steps, timesteps);
#endif
     uint32_t counter = 0;
     saving_steps--;
     
     while(timesteps) {
       timesteps--;
       //printf("main algorithm\n");
       /* *************************************************************************************************************************************** *
	* 																	  *
	* 							ALGORITHM STEP									  *
	* 																	  *
	* *************************************************************************************************************************************** */
       // multiply by Vext propagator (do in callback load) !
       
       // go to momentum space
       //printf("\ntransforming wavefunction to momentum space\n");
       CHECK_CUFFT( hipfftExecZ2Z((global_stuff->plans)[FORWARD_PSI],
				 global_stuff->complex_arr1_dev,
				 global_stuff->complex_arr2_dev,
				 HIPFFT_FORWARD) );
       
       // multiply by T propagator (do in callback) <- ALE KTORY store od FORWARD czy load od INVERSE
       call_kernel_ZZ_1d( ker_propagate, global_stuff->complex_arr2_dev, global_stuff->propagator_T_dev, (global_stuff->streams)[SIMULATION_STREAM] );
       
       
       // count norm using own function
       //call_kernel_ZD_1d( ker_count_norm_wf_1d, global_stuff->complex_arr2_dev, global_stuff->norm_dev,  (global_stuff->streams)[SIMULATION_STREAM], 1024*sizeof(hipDoubleComplex) );
       
       //count norm using CUBLAS       
       //CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, global_stuff->propagator_T_dev, 1, global_stuff->norm_dev) );
       
       
#ifdef DEBUG
       // saving after fft
       HANDLE_ERROR( hipMemcpy(global_stuff->wf_host, global_stuff->complex_arr2_dev, NX*NY*NZ*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );
       
       for (uint64_t ii=0 ; ii < NX*NY*NZ/2; ii++)
         fprintf( (global_stuff->files)[1], "%.15f\t%.15f\t%.15f\n", DKx*ii, creal((global_stuff->wf_host)[ii]), cimag((global_stuff->wf_host)[ii]) );
       for (uint64_t ii = NX*NY*NZ/2 ; ii < NX*NY*NZ; ii++)
         fprintf( (global_stuff->files)[1], "%.15f\t%.15f\t%.15f\n", 2*KxMIN + DKx*ii, creal((global_stuff->wf_host)[ii]), cimag((global_stuff->wf_host)[ii]) );
       
#endif
             
       
       // go back to 'positions`'? space <- JAK JEST PO ANGIELSKU PRZESTRZEN POLOZEN ???
       CHECK_CUFFT( hipfftExecZ2Z((global_stuff->plans)[BACKWARD_PSI],
				 global_stuff->complex_arr2_dev,
				 global_stuff->complex_arr1_dev,
				 HIPFFT_BACKWARD) );
       
       // run kernel to normalize aftter FFT
       call_kernel_Z_1d( ker_normalize_1d, global_stuff->complex_arr1_dev, (global_stuff->streams)[SIMULATION_STREAM] );
       
       
       // evolve in positions` space
       call_kernel_ZZ_1d( ker_propagate, global_stuff->complex_arr1_dev, global_stuff->propagator_Vext_dev, (global_stuff->streams)[SIMULATION_STREAM] );
       
       
       /*
       // count DFT of modulus of wavefunction (in positions` space)
       CHECK_CUFFT( hipfftExecD2Z((global_stuff->plans)[FORWARD_DIPOLAR],
				 global_stuff->double_arr1_dev,
				 global_stuff->complex_arr2_dev) ); // double to complex must be forward, so no need to specify direction
       
       
       
       // count integral in potential of dipolar interactions
       CHECK_CUFFT( hipfftExecZ2Z((global_stuff->plans)[BACKWARD_DIPOLAR],
				 global_stuff->complex_arr2_dev,
				 global_stuff->complex_arr2_dev,
				 HIPFFT_BACKWARD) );
       // normalize (in callback store
       
       // create propagator of Vdip (in)
       */
       
       /* *************************************************************************************************************************************** *
	* 																	  *
	* 							END OF ALGORITHM STEP								  *
	* 																	  *
	* *************************************************************************************************************************************** */
       
       // check norm
       if ( counter == 10000 ) {
	  CHECK_CUBLAS( hipblasDznrm2( cublas_handle, NX*NY*NZ, global_stuff->complex_arr1_dev, 1, global_stuff->norm_dev) );
	  hipDeviceSynchronize();
	  
	  HANDLE_ERROR( hipMemcpyAsync(&norm_host, global_stuff->norm_dev,
				    sizeof(double),
				    hipMemcpyDeviceToHost,
				    (global_stuff->streams)[HELPER_STREAM]) );
	  hipDeviceSynchronize();
	  norm_host *= sqrt(DX);
	  fprintf( (global_stuff->files)[3], "%lu.\t%.15f\n", (timesteps_tot-timesteps)*(saving_tot-saving_steps), norm_host );
	  counter = 0;
       }
       counter++;
       
     }
     
     
     
       // saving after ifft
       HANDLE_ERROR( hipMemcpy(global_stuff->wf_host, global_stuff->complex_arr1_dev, NX*NY*NZ*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );
       for (uint64_t ii=0 ; ii < NX*NY*NZ; ii++)
         fprintf( (global_stuff->files)[2], "%.15f\t%.15f\t%.15f\n", XMIN+DX*ii, creal((global_stuff->wf_host)[ii]), cimag((global_stuff->wf_host)[ii]) );
#ifdef DEBUG     
       call_kernel_Z_1d( ker_print_Z, global_stuff->complex_arr1_dev, (global_stuff->streams)[SIMULATION_STREAM] );
#endif
     if (!saving_steps) FLAG_RUN_SIMULATION = false;
  }
//#ifdef DEBUG
//#endif
  
#ifdef DEBUG
  printf("last barrier reached by %s.\n",thread_names[SIMULATION_THRD]);
#endif
  hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
  
  // free memory on host
  HANDLE_ERROR( hipHostFree(global_stuff->wf_host) );
#ifdef DEBUG
  HANDLE_ERROR( hipHostFree(propagator_T_host) );
#endif
  /*
  /// free memory on device
  HANDLE_ERROR( hipFree(global_stuff->complex_arr1_dev) ); 	//
  HANDLE_ERROR( hipFree(global_stuff->complex_arr2_dev) ); 	//
  //HANDLE_ERROR( hipFree(global_stuff->double_arr1_dev)  ); 	//
  HANDLE_ERROR( hipFree(global_stuff->propagator_T_dev) ); 	//
  //HANDLE_ERROR( hipFree(global_stuff->propagator_Vext_dev) );	//
  //HANDLE_ERROR( hipFree(global_stuff->Vdip_dev) );		//
  
  
  //HANDLE_ERROR( hipFree(global_stuff->mean_T_dev) ); // result of integral with kinetic energy operator in momentum representaion
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vdip_dev) ); // result of integral with Vdip operator in positions' representation
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vext_dev) ); // result of integral with Vext operator in positions' representation
  //HANDLE_ERROR( hipFree(global_stuff->mean_Vcon_dev) ); // result of integral with Vcon operator in positions' representation
  HANDLE_ERROR( hipFree(global_stuff->norm_dev) ); //
  */
  pthread_barrier_wait (&barrier_global);
  pthread_exit(NULL);
}





/* ************************************************************************************************************************************* *
 * 																	 *
 * 							HELPER THREAD									 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * - allocation memory on device
 * - allocation plans
 * 
 */
void* helper_thread(void* passing_ptr) {
  
  //stick_this_thread_to_core(2);
  
  pthread_barrier_wait (&barrier_global);
  printf("running %s thread.\n",thread_names[HELPER_THRD]);
  
  // init memory on device
  // arrays for wavefunction
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->complex_arr1_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	//
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->complex_arr2_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	//
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->double_arr1_dev), sizeof(double) * NX*NY*NZ) );		//
  
  // constant arrays
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->propagator_T_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) ); 	// array of constant factors e^-i*k**2/2*dt
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->propagator_Vext_dev), sizeof(hipDoubleComplex) * NX*NY*NZ) );// array of constant factors e^-i*Vext*dt
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->Vdip_dev), sizeof(double) * NX*NY*NZ) ); 			// array of costant factors <- count on host with spec funcs lib or use Abramowitz & Stegun approximation
  
  // scalar variables
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->mean_T_dev), sizeof(double))    ); // result of integral with kinetic energy operator in momentum representaion
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->mean_Vdip_dev), sizeof(double)) ); // result of integral with Vdip operator in positions' representation
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->mean_Vext_dev), sizeof(double)) ); // result of integral with Vext operator in positions' representation
  //HANDLE_ERROR( hipMalloc((void**) &(global_stuff->mean_Vcon_dev), sizeof(double)) ); // result of integral with Vcon operator in positions' representation
  HANDLE_ERROR( hipMalloc((void**) &(global_stuff->norm_dev), sizeof(double)) ); // variable to hold norm of wavefunction
  
  
#ifdef DEBUG
  printf("allocated memory on device.\n");
#endif
  
  for (uint8_t ii = 0; ii < num_streams; ii++)
    HANDLE_ERROR(	hipStreamCreate( &(global_stuff->streams[ii]) )	);
  
#ifdef DEBUG
  printf("1st barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  //hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
  
  // creating plans with callbacks
  global_stuff->plans = (hipfftHandle*) malloc( (size_t) sizeof(hipfftHandle)*num_plans );
#ifdef DEBUG
  printf("array of plans allocated.\n");
#endif
  for (uint8_t ii = 0; ii < num_plans; ii++) {
    CHECK_CUFFT(  hipfftCreate( (global_stuff->plans)+ii )  ); // allocates expandable plans
    //printf("%d\n",(global_stuff->plans)[ii]);
  }
  
#ifdef DEBUG
  printf("expandable plans allocated.\n");
#endif
  
  size_t work_size; // CHYBA TO MUSI BYC TABLICA !!!
#if (DIM == 1)
  printf("creating CUFFT plans in 1d case.\n");
  // wavefunction forward
  // hipfftMakePlan1d(plan, N, HIPFFT_Z2Z, 1, &work_size);
  CHECK_CUFFT( hipfftMakePlan1d( (global_stuff->plans)[FORWARD_PSI], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size ) 	);
#ifdef DEBUG
  //pthread_barrier_wait (&barrier);
#endif
  // associate transform with specified stream
  
  // wavefunction forward
  CHECK_CUFFT( hipfftSetStream(  (global_stuff->plans)[FORWARD_PSI], (global_stuff->streams)[SIMULATION_STREAM] ) );
  //printf("%d\n",(global_stuff->plans)[FORWARD_PSI]);
  
  // wavefunction inverse
  //  printf("%p\n",(global_stuff->plans)+BACKWARD_PSI);
  CHECK_CUFFT( hipfftMakePlan1d( (global_stuff->plans)[BACKWARD_PSI], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (global_stuff->plans)[BACKWARD_PSI], (global_stuff->streams)[SIMULATION_STREAM]) );
  //printf("%d\n",(global_stuff->plans)[BACKWARD_PSI]);
  
  // modulus of wavefunction forward
  CHECK_CUFFT( hipfftMakePlan1d( (global_stuff->plans)[FORWARD_DIPOLAR], NX*NY*NZ, HIPFFT_D2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (global_stuff->plans)[FORWARD_DIPOLAR], (global_stuff->streams)[HELPER_STREAM] ) );
  //printf("%d\n",(global_stuff->plans)[FORWARD_DIPOLAR]);
  
  // integral in potential of dipolar inteaction
  CHECK_CUFFT( hipfftMakePlan1d( (global_stuff->plans)[BACKWARD_DIPOLAR], NX*NY*NZ, HIPFFT_Z2Z, 1, &work_size )	);
  CHECK_CUFFT( hipfftSetStream(  (global_stuff->plans)[BACKWARD_DIPOLAR], (global_stuff->streams)[HELPER_STREAM]) ); // WLASCIWIE TUTAJ NIE WIADOMO W KTORYM STREAMIE?
  //printf("%d\n",(global_stuff->plans)[BACKWARD_DIPOLAR]);
  
#elif (DIM == 2)
  
#elif (DIM == 3)
  
#endif // case DIM for plan
  printf("\tplans created\n");
  
  // !!! SPRAWDZIC !!! funckje: <- co robia?
  //hipfftResult hipfftSetAutoAllocation(hipfftHandle *plan, bool autoAllocate);
  //cufftSetCompatibilityMode() <- musi byc wywolana po create a przed make plan
  
  
  /* ************************************
   * 			CUBLAS		*
   * ************************************/
  
  CHECK_CUBLAS( hipblasCreate(&cublas_handle) );
  CHECK_CUBLAS( hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) ); // this means we can use only device pointers to scalars (required by CUBLAS routines)
  
  printf("CUBLAS initialized!\n");
  
  
  
  
  
  
#ifdef DEBUG
  printf("created FFT plans.\n");
#endif
#ifdef DEBUG
  printf("2nd barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  pthread_barrier_wait (&barrier);
  
  
  
  // start algorithm
  // dt =
  /*uint16_t timesteps;
  while( FLAG_RUN_SIMULATION ) { // simulation will be runing until the flag is set to false
#ifdef DEBUG
     timesteps = 1;
#else
     timesteps = 1000;
#endif
     while(timesteps) {
       timesteps--;
       
       //FLAG_RUN_SIMULATION = false;
     }
  }*/
  
  //hipDeviceSynchronize();
  pthread_barrier_wait (&barrier);
#ifdef DEBUG
  printf("last barrier reached by %s.\n",thread_names[HELPER_THRD]);
#endif
  
  CHECK_CUBLAS( hipblasDestroy(cublas_handle) );
  //CHECK_CUBLAS( cublasShutdown() );
  
  
  
  
  for (uint8_t ii = 0; ii < num_streams; ii++)
    HANDLE_ERROR(	hipStreamDestroy( (global_stuff->streams[ii]) )	);
  
  pthread_barrier_wait (&barrier_global);
  pthread_exit(NULL);
}


void alloc_device(){
  
  
}


void alloc_host() {
  
  // must use
  
}
