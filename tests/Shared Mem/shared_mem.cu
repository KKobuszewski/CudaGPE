#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void staticReverse(hipDoubleComplex *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(hipDoubleComplex *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  hipDoubleComplex a[n], r[n], d[n];
  
  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }
  
  hipDoubleComplex *d_d;
  hipMalloc(&d_d, n * sizeof(hipDoubleComplex)); 
  
  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  staticReverse<<<1,n>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
  
  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  
  // WYTESTOWAC odpalenie tego kernela z parametrami dimGrid(512,1,1)/dimBlocks(128,1,1)/3KB = 3*(1<<10) KB Shared Mem per thread !!!
  printf("sizeof hipDoubleComplex: %d",sizeof(hipDoubleComplex));
  printf("memory needed: %d",n*sizeof(hipDoubleComplex));
  dynamicReverse<<<1,n,n*sizeof(hipDoubleComplex)>>>(d_d, n);
  
  
  
  hipMemcpy(d, d_d, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
}