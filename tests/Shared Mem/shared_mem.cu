#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void staticReverse(hipDoubleComplex *d, int n)
{
  __shared__ hipDoubleComplex s[64];
  hipDoubleComplex t = make_hipDoubleComplex( (double)threadIdx.x, (double)(n-threadIdx.x-1) );
  hipDoubleComplex tr = make_hipDoubleComplex( (double)(n-threadIdx.x-1), (double)threadIdx.x );
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(hipDoubleComplex *d, int n)
{
  extern __shared__ hipDoubleComplex s[];
  hipDoubleComplex t = make_hipDoubleComplex(threadIdx.x, n-threadIdx.x-1);
  hipDoubleComplex tr = make_hipDoubleComplex(n-t-1, threadIdx);
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  hipDoubleComplex a[n], r[n], d[n];
  
  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }
  
  hipDoubleComplex *d_d;
  hipMalloc(&d_d, n * sizeof(hipDoubleComplex)); 
  
  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  staticReverse<<<1,n>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
  
  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  
  // WYTESTOWAC odpalenie tego kernela z parametrami dimGrid(512,1,1)/dimBlocks(128,1,1)/3KB = 3*(1<<10) KB Shared Mem per thread !!!
  printf("sizeof hipDoubleComplex: %d",sizeof(hipDoubleComplex));
  printf("memory needed: %d",n*sizeof(hipDoubleComplex));
  dynamicReverse<<<1,n,n*sizeof(hipDoubleComplex)>>>(d_d, n);
  
  
  
  hipMemcpy(d, d_d, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
}