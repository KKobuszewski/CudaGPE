#include  <hipfft/hipfft.h> 


#define  NX  256 
#define  BATCH  10 



void cuda_dft_1d() {
  
  hipfftHandle plan; 
  hipfftComplex *data; 
  
  hipMalloc((void**)&data,  sizeof(hipfftComplex)*NX*BATCH );
  
  /* Create   a  1D  FFT  plan. */ 
  hipfftPlan1d(&plan, NX, CUFFT C2C, BATCH);
  
  /* Use  the  CUFFT  plan  to  transform  the  signal  in place. */ 
  hipfftExecC2C(plan, data, data, CUFFT FORWARD); 
  
  /* Destroy  the  CUFFT  plan. */ 
  hipfftDestroy(plan); 
  hipFree(data);
}