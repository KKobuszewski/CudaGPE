#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cudautils.cuh"


extern hipEvent_t start;
extern hipEvent_t stop;

double print_cudatimeit(const char* message) {
  float computationTime;
  hipEventElapsedTime(&computationTime, start, stop);
  printf( "time of %-60s %lf s\n", message, (double) computationTime );
  return (double) computationTime;
}

double fprint_cudatimeit(FILE* file) {
  float computationTime;
  hipEventElapsedTime(&computationTime, start, stop);
  fwrite(&computationTime, (size_t) sizeof(double), 1, file);
  return (double) computationTime;
}

// void cudaCheckErrors(hipError_t err,const char* action){
//   if (hipGetLastError() != hipSuccess)
//   {
//         fprintf(stderr, "Failed to:<< %s >>(error code %s)!\n", action, hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//   }
// }