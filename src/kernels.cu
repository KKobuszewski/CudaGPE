#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <pthread.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "global.h"
#include "kernels.cuh"
#include "cuda_complex_ext.cuh"





/* ************************************************************************************************************************************* *
 * 																	 *
 * 							KERNELS TYPE Z									 *
 * 																	 *
 * ************************************************************************************************************************************* */

#define SIGMA ( XMAX*sqrt(2./(3.14159265358979323846*NX)) )
#ifndef IMAG_TIME
    #define OFFSET_X ((double) 0.1)
#else
    #define OFFSET_X ((double) 0.0)
#endif
__global__ void ker_gauss_1d(hipDoubleComplex* data) {
  // get the index of thread
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  const uint64_t N = NX*NY*NZ;
  
  // allocate constants in shared memory
  //const double x0 = (-5*SIGMA);
  //const double dx = (10*SIGMA)/((double) N);
  
  if (ii < N) {
    data[ii] = make_hipDoubleComplex( sqrt(INV_SQRT_2PI/SIGMA)*exp(-(XMIN + ii*DX + OFFSET_X)*(XMIN + ii*DX + OFFSET_X)/4/(SIGMA*SIGMA)), 0. );
  }
  
  __syncthreads();
  if ( ii == 0)
    printf("sigma: %.15f\n",SIGMA);
}


__global__ void ker_const_1d(hipDoubleComplex* wf) {
  // get the index of thread
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
    
  while (ii < NX) {
    wf[ii] = make_hipDoubleComplex( (1./(XMAX-XMIN)), 0. );
    ii += blockDim.x * gridDim.x;
  }
}


/*
 * Divides the result of inverse cufft by number of samples (to get unitary form of DFT).
 * 
 * NOTE: hipfftDoubleComplex is just typdef for hipDoubleComplex ! (no need to include <hip/hip_complex.h> if only cufft necessary)
 * 	 here used to associate it with inverse cufft
 */
__global__ void ker_normalize_1d(hipfftDoubleComplex* cufft_inverse_data) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // in both kernel as well as callback we use predefined N to have comparable performance results
  
  while (ii < NX) {
    cufft_inverse_data[ii] = make_hipDoubleComplex( hipCreal(cufft_inverse_data[ii])/((double) NX), hipCimag(cufft_inverse_data[ii])/((double) NX) );
    // check division Intrinsics ddiv_rz <- round to zero mode (maybe less problems with norm ??? & faster )
    //cufft_inverse_data[ii] = make_hipDoubleComplex( __ddiv_rn(hipCreal(cufft_inverse_data[ii]),(double) NX) ,
	//					   __ddiv_rn(hipCimag(cufft_inverse_data[ii]),(double) NX) );
    ii += blockDim.x * gridDim.x;
  }
}

// dla funckji roznych dla innych DIM mozna zrobic makro wybierajace odpowiednia funckja lub makro 'krojace funcje' na opcje w zaleznosci od wymiaru
__global__ void ker_create_propagator_T(hipDoubleComplex* propagator_T_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  // tutaj mozna sporbowac uzyc shared memory na k !
  
  // CASE DIM 1D
#if (DIM == 1)
  if (ii < NX*NY*NZ/2) {
    // range [0, KMAX]
    const double kx_ii = DKx*ii;
    propagator_T_dev[ii] = make_hipDoubleComplex( cos(kx_ii*kx_ii*DT/2),-sin(kx_ii*kx_ii*DT/2)  ); // array of constants e^(-I*k^2/2*dt) = cos( -kx^2/2dt ) + I*sin( kx^2/2dt ) = cos( +kx^2/2dt ) - I*sin( +kx^2/2dt )
    
    // make sure that tramsform will be unitary
    /*propagator_T_dev[ii] = make_hipDoubleComplex( hipCreal(propagator_T_dev[ii]) / hipCabs(propagator_T_dev[ii]),
						 hipCimag(propagator_T_dev[ii]) / hipCabs(propagator_T_dev[ii]) );*/
    
  }
  else if (ii < NX*NY*NZ) {
    // range [KMIN = -KMAX, -DK]
    const double kx_ii = 2*KxMIN + DKx*ii;
    propagator_T_dev[ii] = make_hipDoubleComplex( cos(kx_ii*kx_ii*DT/2),-sin(kx_ii*kx_ii*DT/2)  );
#ifdef DEBUG
    if (ii < NX*NY*NZ/2) printf("\nError in kernel creating propagator T!\tWrong index in 'higher part' of FFT.\n");
#endif
  }
  
  // CASE DIM 2D
#elif (DIM == 2)
  
  // CASE DIM 3D
#elif (DIM == 3)


#endif // case dimension for propagator T
  
}


__global__ void ker_print_Z(hipDoubleComplex* arr_dev)
{
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  ii *= 32;
  while (ii < NX) {
    printf("%lu\t%.15f + %.15fj\t%.15f * exp( j*%.15f )\n", ii, hipCreal(arr_dev[ii]), hipCimag(arr_dev[ii]), hipCabs(arr_dev[ii]), cuCarg(arr_dev[ii]) );
    ii += blockDim.x * gridDim.x;
  }
}


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							KERNELS TYPE ZD									 *
 * 																	 *
 * ************************************************************************************************************************************* */

__global__ void ker_multiplyZD(hipDoubleComplex* complex_arr_dev, double* double_arr_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    // WYTESTOWAC CZY SZYBSZE NIE BEDZIE OBLICZANIE PROPAGATORA
    complex_arr_dev[ii] = hipCmul( complex_arr_dev[ii], double_arr_dev[ii] );
    ii += blockDim.x * gridDim.x;
  }
}

__global__ void ker_modulus_wf_1d(hipDoubleComplex* complex_arr_dev, double* double_arr_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX) {
    //double_arr_dev = hipCabs(complex_arr_dev[ii])*hipCabs(complex_arr_dev[ii]);
    double_arr_dev[ii] = sqrt( hipCreal(complex_arr_dev[ii])*hipCreal(complex_arr_dev[ii]) + hipCimag(complex_arr_dev[ii])*hipCimag(complex_arr_dev[ii]) );
    ii += blockDim.x * gridDim.x;
  }
  
}

__global__ void ker_modulus_pow2_wf_1d(hipDoubleComplex* complex_arr_dev, double* double_arr_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
#ifdef DEBUG
  if(ii%4 == 0) printf("x:%.15f\twf:%15f + %15fj\tRe^2:%15f\tIm^2:%15f\n", XMIN + ii*DX, complex_arr_dev[ii].x, complex_arr_dev[ii].y, complex_arr_dev[ii].x*complex_arr_dev[ii].x, complex_arr_dev[ii].y*complex_arr_dev[ii].y);
#endif
  
  while (ii < NX) {
    //double_arr_dev = hipCabs(complex_arr_dev[ii])*hipCabs(complex_arr_dev[ii]);
    double_arr_dev[ii] = complex_arr_dev[ii].x*complex_arr_dev[ii].x + complex_arr_dev[ii].y*complex_arr_dev[ii].y;
    ii += blockDim.x * gridDim.x;
  }
  
}

__global__ void ker_arg_wf_1d(hipDoubleComplex* complex_arr_dev, double* double_arr_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (ii < NX*NY*NZ) {
    //double_arr_dev[ii] = atan2( hipCimag(complex_arr_dev[ii]), hipCreal(complex_arr_dev[ii]) ); // in case line below doesn't work
    double_arr_dev[ii] = cuCarg(complex_arr_dev[ii]); // this function is declared in cuda_complex_ext.cuh
  }
  
}

/* 
 * TEN KERNEL POWINIEN BYC JESZCZE UDOSKONALONY JAK KAZDY Z RESZTA
 * NA RAZIE MOZNA ZASTAPIC CUBLAS: hipblasStatus_t hipblasDznrm2(hipblasHandle_t handle, int n,const hipDoubleComplex *x, int incx, double *result)
 * http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-nrm2
 */
__global__ void ker_count_norm_wf_1d(hipDoubleComplex* complex_arr_dev, double* norm_dev) {
  extern __shared__ double shared_mods[]; // CZY TO SIE ZMIESCI W SHARED MEMORY ???
  
  uint16_t tid = threadIdx.x;
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  if (ii == 0) *norm_dev = 0;
  
  // load |psi|^2(x) to shared memory
  //shared_mods[tid] = hipCreal(complex_arr_dev[ii])*hipCreal(complex_arr_dev[ii]) + hipCimag(complex_arr_dev[ii])*hipCimag(complex_arr_dev[ii]);
  if (ii < NX*NY*NZ)
    shared_mods[tid] = cuCSqAbs(complex_arr_dev[ii]);// + cuCSqAbs(complex_arr_dev[ii + blockDim.x]); // blockDim.x MUSI BYC ODPOWIEDNIEJ DLUGOSCI
  __syncthreads();
  
  // simple reduction - look at http://sbel.wisc.edu/Courses/ME964/2012/Lectures/lecture0313.pdf
  for (uint32_t s=blockDim.x/2; s > 0; s>>=1) {
    // sequential addressing in shared memory
    if (tid < s) {
      shared_mods[tid] += shared_mods[tid+s];
    }
    
    __syncthreads();
  }
  
  // add results to variable in global memory <- CZY W TEN SPOSOB TO NIE BEDZIE POWODOWAC BLEDOW ???
  if (tid==0) *norm_dev += shared_mods[0];
  
  __syncthreads();
  //if (ii == 0) *norm_dev *= sqrt(DX);
  
}

__global__ void ker_normalize_1d(hipDoubleComplex* data, double* norm) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // SPRAWDZIC CZY NIE DA SIE PRZYSPIESZYC POPRZEZ CONSTANT / SHARED MEMOMRY (SKOPIOWAC TAM WARTOSC NORMY) !!!
  
  while (ii < NX) {
    data[ii] = make_hipDoubleComplex( hipCreal(data[ii])/(*norm)/sqrt(DX), hipCimag(data[ii])/(*norm)/sqrt(DX) );
    ii += blockDim.x * gridDim.x;
  }
}


__global__ void ker_energy_T_1d(hipDoubleComplex* wf_k, double* T_mean) {
  extern __shared__ double shared_mods[]; // CZY TO SIE ZMIESCI W SHARED MEMORY ???
  
  uint16_t tid = threadIdx.x;
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  if (ii == 0) *T_mean = 0;
  
  // load psi* x Op(psi) to shared memory;
  if (ii < NX*NY*NZ)
    shared_mods[tid] = operator_T_dev(wf_k[ii], ii);
  __syncthreads();
  
  // simple reduction - look at http://sbel.wisc.edu/Courses/ME964/2012/Lectures/lecture0313.pdf
  for (uint32_t s=blockDim.x/2; s > 0; s>>=1) {
    // sequential addressing in shared memory
    if (tid < s) {
      shared_mods[tid] += shared_mods[tid+s];
    }
    
    __syncthreads();
  }
  
  // add results to variable in global memory <- CZY W TEN SPOSOB TO NIE BEDZIE POWODOWAC BLEDOW ???
  if (tid==0) *T_mean += shared_mods[0];
}

__global__ void ker_energy_Vext_1d(hipDoubleComplex* wf, double* Vext_mean) {
  extern __shared__ double shared_mods[]; // CZY TO SIE ZMIESCI W SHARED MEMORY ???
  
  uint16_t tid = threadIdx.x;
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  if (ii == 0) *Vext_mean = 0;
  
  // load psi* x Op(psi) to shared memory;
  if (ii < NX*NY*NZ)
    shared_mods[tid] = operator_Vext_dev(wf[ii], ii);
  __syncthreads();
  
  // simple reduction - look at http://sbel.wisc.edu/Courses/ME964/2012/Lectures/lecture0313.pdf
  for (uint32_t s=blockDim.x/2; s > 0; s>>=1) {
    // sequential addressing in shared memory
    if (tid < s) {
      shared_mods[tid] += shared_mods[tid+s];
    }
    
    __syncthreads();
  }
  
  // add results to variable in global memory <- CZY W TEN SPOSOB TO NIE BEDZIE POWODOWAC BLEDOW ???
  if (tid==0) *Vext_mean += shared_mods[0];
}



//		!!!!!!!!!!!!!!!!!!   TO NIE DZIALA   !!!!!!!!!!!!!!!!!!!
/*
 * Kernel that counts expected value of an operator represented by function passed in dev_funcZ_ptr_t operator
 * dev_funcZ_ptr_t operator - (host copy of) device pointer to device function representing action of diagonal operator on wavefunction
 * double* mean - pointer to device memory location to store <operator>
 * 
 */
__global__ void ker_operator_mean_1d( dev_funcZ_ptr_t func, hipDoubleComplex* wf, double* mean ) {
  extern __shared__ double shared_mods[]; // CZY TO SIE ZMIESCI W SHARED MEMORY ???
  
  uint16_t tid = threadIdx.x;
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // load psi* x Op(psi) to shared memory;
  if (ii < NX*NY*NZ)
    shared_mods[tid] = func(wf[ii], ii);
  __syncthreads();
  
  // simple reduction - look at http://sbel.wisc.edu/Courses/ME964/2012/Lectures/lecture0313.pdf
  for (uint32_t s=blockDim.x/2; s > 0; s>>=1) {
    // sequential addressing in shared memory
    if (tid < s) {
      shared_mods[tid] += shared_mods[tid+s];
    }
    
    __syncthreads();
  }
  
  // add results to variable in global memory <- CZY W TEN SPOSOB TO NIE BEDZIE POWODOWAC BLEDOW ???
  if (tid==0) *mean += shared_mods[0];
  
  //__syncthreads();
  //if (ii == 0) *mean *= sqrt(DX);
    
}


/*
 * PROPAGATION VIA CONTACT INTERACTIONS
 */
__global__ void ker_propagate_Vcon_1d(hipDoubleComplex* wf, double* density) {
  //extern __shared__ double factor[];
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  //uint16_t tid = threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    
    //factor[tid] = G_CONTACT*density[ii]*NX*NY*NZ;// gN|\psi|^2
    
// #ifdef DEBUG
//     if (ii%4 == 0) printf("x: %.15f\twavefunction before progration Vcon: %.15f + %.15fj\tdensity: %.15f\n", XMIN + ii*DX,hipCreal(wf[ii]),hipCimag(wf[ii]),density[ii]);
//     __syncthreads();
// #endif
    // WYTESTOWAC CZY SZYBSZE NIE BEDZIE OBLICZANIE PROPAGATORA
#ifdef REAL_TIME
    wf[ii] = hipCmul(  wf[ii], make_hipDoubleComplex(  cos( G_CONTACT*density[ii]*DT ),-sin( G_CONTACT*density[ii]*DT )  )  );
    //wf[ii] = hipCmul(  wf[ii], make_hipDoubleComplex(cos(factor[tid]*DT),-sin(factor[tid]*DT))  );
#endif
#ifdef IMAG_TIME
    wf[ii] = hipCmul( wf[ii], exp(-G_CONTACT*density[ii]*DT) );
#endif
// #ifdef DEBUG
//     if (ii < 10) printf("wavefunction after progration Vcon: %.15f + %.15fj\n",hipCreal(wf[ii]),hipCimag(wf[ii]));
// #endif
    
    ii += blockDim.x * gridDim.x;
  }
}


/*
 * PROPAGATION VIA INTERACTIONS (with predefined interactions` potential)
 * 
 * (this evolution is made in positions` space)
 * hipDoubleComplex* wf - wavefunction
 * hipDoubleComplex* Vint - interactions` potential (integral is counted via convolution with FFT)
 */
__global__ void ker_propagate_Vint_1d(hipDoubleComplex* wf, hipDoubleComplex* Vint) {
  //extern __shared__ double factor[];
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  double Re_Vint = Vint[ii].x;
  double Im_Vint = Vint[ii].y;
  
  while (ii < NX*NY*NZ) {
    
    //factor[tid] = G_CONTACT*density[ii]*NX*NY*NZ;// gN|\psi|^2
    
#ifdef DEBUG
    if (Im_Vint > 0) printf("x: %.15f\twavefunction before progration Vint: %.15f + %.15fj\tVint: %.15f + %.15fj\n", XMIN + ii*DX,hipCreal(wf[ii]),hipCimag(wf[ii]),Re_Vint,Im_Vint);
    __syncthreads();
#endif
#ifdef REAL_TIME
    // TODO: Check if taking imaginary part is good? 
    wf[ii] = hipCmul(  wf[ii], make_hipDoubleComplex(  exp(Im_Vint*DT)*cos( Re_Vint*DT ),-exp(Im_Vint*DT)*sin( Re_Vint*DT )  )  );
#endif
#ifdef IMAG_TIME
    wf[ii] = hipCmul( wf[ii], make_hipDoubleComplex(  exp(-Re_Vint*DT)*cos( Re_Vint*DT ),-exp(Im_Vint)*sin( Re_Vint*DT )  ) );
#endif
#ifdef DEBUG
    if (ii < 10) printf("wavefunction after progration Vcon: %.15f + %.15fj\tVint: %.15f + %.15fj\n", XMIN + ii*DX,hipCreal(wf[ii]),hipCimag(wf[ii]),Re_Vint,Im_Vint);
#endif
    
    ii += blockDim.x * gridDim.x;
  }
}

/*
 *          PHASE IMPRINTING
 * double* phase is an array contaning phase on grid in radians
 */
__global__ void ker_phase_imprint_1d(hipDoubleComplex* wf, double* phase) {
    uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
    
    // register variables <- quicker?
    double mod_wf;
    double phase_reg;
    
    while (ii < NX*NY*NZ) {
        mod_wf = hipCabs(wf[ii]);
        phase_reg = phase[ii];
        
        wf[ii] = make_hipDoubleComplex( mod_wf*cos(phase_reg) , mod_wf*sin(phase_reg) );
        
        ii += blockDim.x * gridDim.x;
    }
    
}





/* ************************************************************************************************************************************* *
 * 																	 *
 * 							KERNELS TYPE ZZ									 *
 * 																	 *
 * ************************************************************************************************************************************* */

/*
 * Element-wise vector multiplication
 */
__global__ void ker_multiplyZZ(hipDoubleComplex* wf_momentum_dev, hipDoubleComplex* propagator_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    // WYTESTOWAC CZY SZYBSZE NIE BEDZIE OBLICZANIE PROPAGATORA
    wf_momentum_dev[ii] = hipCmul( wf_momentum_dev[ii], propagator_dev[ii] );
    ii += blockDim.x * gridDim.x;
  }
}

/*
 * Element-wise multiplication
 * Multiply elements of first array by real parts of elements of second array and saves result to second array
 */
__global__ void ker_multiplyZReZ(hipDoubleComplex* complex_arr1_dev, hipDoubleComplex* complex_arr2_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    double factor = complex_arr2_dev[ii].x;
    complex_arr2_dev[ii] = hipCmul( complex_arr1_dev[ii], factor );
    ii += blockDim.x * gridDim.x;
  }
}


__global__ void ker_T_wf(hipDoubleComplex* wf_momentum_dev, hipDoubleComplex* result_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    result_dev[ii] = hipCmul(wf_momentum_dev[ii], kx_dev(ii)*kx_dev(ii));
    ii += blockDim.x * gridDim.x;
  }
  
}

__global__ void ker_Vext_wf(hipDoubleComplex* wf_dev, hipDoubleComplex* result_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    result_dev[ii] = hipCmul(wf_dev[ii], (XMIN + ii*DX)*(XMIN + DX*ii) );
    ii += blockDim.x * gridDim.x;
  }
  
}


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							KERNELS TYPE ZDZ									 *
 * 																	 *
 * ************************************************************************************************************************************* */


__global__ void ker_Vcon_wf(hipDoubleComplex* wf_dev, double* density, hipDoubleComplex* result_dev) {
  uint64_t ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  while (ii < NX*NY*NZ) {
    result_dev[ii] = hipCmul(wf_dev[ii], G_CONTACT*density[ii] );
    ii += blockDim.x * gridDim.x;
  }
  
}


// cross sections of wavefunction
__global__ void get_cross_sectionX();
__global__ void get_cross_sectionY();
__global__ void get_cross_sectionZ();
__global__ void get_cross_sectionXY();
__global__ void get_cross_sectionXZ();
__global__ void get_cross_sectionYZ();


// integrals
__global__ void mean_kinetic_energy();
__global__ void mean_potential_energy();
__global__ void mean_contact_interaction_energy();
__global__ void mean_dipolar_interaction_energy();
__global__ void mean_momentum();
__global__ void get_norm();
// angular momnetum???

// DEVICE ONLY FUNCTIONS

static __device__ void normalize();
//__device__



// callbacks
//static __device__ hipDoubleComplex propagate_Vext();
//static __device__ hipDoubleComplex propagate_T();
//static __device__ hipDoubleComplex propagate_Vcon();
static __device__ hipDoubleComplex propagate_Vdip();

// pointer to callbacks' functions
__device__ hipfftCallbackLoadZ CB_LD_MOMENTUM_SPACE_FORWARD();
__device__ hipfftCallbackStoreZ CB_ST_MOMENTUM_SPACE_FORWARD();
__device__ hipfftCallbackLoadZ CB_LD_MOMENTUM_SPACE_INVERSE();
__device__ hipfftCallbackStoreZ CB_ST_MOMENTUM_SPACE_INVERSE();
__device__ hipfftCallbackLoadD CB_LD_DIPOLAR_FORWARD();
__device__ hipfftCallbackStoreZ CB_ST_DIPOLAR_FORWARD();
__device__ hipfftCallbackLoadZ CB_LD_DIPOLAR_INVERSE();
__device__ hipfftCallbackStoreZ CB_ST_DIPOLAR_INVERSE();


// tests with callbacks


/* ************************************************************************************************************************************* *
 * 																	 *
 * 							LOAD CALLBACKS									 *
 * 																	 *
 * ************************************************************************************************************************************* */

static __device__ hipfftDoubleComplex cufftSgn(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
    if (offset < (NX*NY*NZ)/2)
      return make_hipDoubleComplex(-1.,0.);
    else
      return make_hipDoubleComplex(1.,0.);
}

static __device__ hipfftDoubleComplex cufftGauss_1d(void *dataIn, 
						  size_t offset, 
						  void *callerInfo, 
						  void *sharedPtr) 
{
  // allocate constants in shared memory <- how to do that???
  const double x0 = (-5*SIGMA);
  const double dx = (10*SIGMA)/((double) NX*NY*NZ);
  return make_hipDoubleComplex( sqrt(INV_SQRT_2PI/SIGMA)*exp(-(x0 + offset*dx)*(x0 + offset*dx)/4/(SIGMA*SIGMA)), 0. );
}



