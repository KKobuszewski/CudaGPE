#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <cudaProfiler.h>

#include "cuda_dft.cuh"


__global__ void cudaGauss_1d(hipfftDoubleComplex* data, const uint64_t N) {
  // get the index of thread
  int ii = blockIdx.x*blockDim.x + threadIdx.x;
  
  // allocate constants in shared memory
  const double x0 = (-5*SIGMA);
  const double dx = (10*SIGMA)/((double) N);
  
  if (ii < N)
    data[ii] = make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + ii*dx)*(x0 + ii*dx)/2/SIGMA)/SIGMA, 0.);//GAUSSIAN
}


void cuda_dft_1d(const uint64_t N, FILE** array_timing) {
   
  // initilizing files to save data
  const uint8_t filename_str_lenght = 128;
  const uint8_t dim = 1;
  
  // co jest??
  double  *data_dev, *data_host; // UWAGA - SMIESZNIE SIE ALOKUJE PAMIEC!!!
  
  char filename1d[filename_str_lenght];
  sprintf(filename1d,"./data/cufft_%dd_N%lu.bin",dim,N );
  printf("1d cufft example save in: %s\n",filename1d);
  FILE *file1d = fopen(filename1d, "wb");
  if (file1d == NULL)
  {
      printf("Error opening file %s!\n",filename1d);
      exit(EXIT_FAILURE);
  }
  
  checkCudaErrors( hipMalloc( (void**)&data_dev,  sizeof(hipfftDoubleComplex)*N ) );
  checkCudaErrors( hipHostAlloc( (void**)&data_host, sizeof(hipfftDoubleComplex)*N ,hipHostMallocDefault) );
  
  // fill array
  int threadsPerBlock = 512;
  dim3 dimGrid( (N + threadsPerBlock - 1)/threadsPerBlock, 1, 1 ); // (numElements + threadsPerBlock - 1) / threadsPerBlock
  dim3 dimBlock(threadsPerBlock,1,1);
  checkCudaErrors( cudaGauss_1d<< dimGrid,dimBlock >>(data_dev) );
  
  checkCudaErrors( hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );
  hipDeviceSynchronize();
  for (uint16_t ii = 0; ii < N; ii++) {
    //fprintf(file1d,"",creal(data_host[ii]),cimag(data_host[ii]), sizeof(hipfftDoubleComplex));
    fwrite(data_host+2*ii, sizeof(hipfftDoubleComplex),1,file1d);
  }
  
  
  /* Create   a  1D  FFT  plan. */
  //cufftSetCompatibilityMode(CUFFT_COMPATIBILITY_NATIVE);
  //hipfftHandle  plan; 
  //hipfftPlan1d(&plan, NX, CUFFT Z2Z,1); // we want to make transform of only one 1D array
  
  /* Use  the  CUFFT  plan  to  transform  the  signal  in place. */ 
  //HANDLE_ERROR( hipfftExecZ2Z(plan, data_dev, data_dev, CUFFT FORWARD) ); 
  
  
  
  
  //  cleaning up the mesh
  //HANDLE_ERROR( hipfftDestroy(plan) ); 
  checkCudaErrors( hipFree(data_dev) );
  checkCudaErrors( hipHostFree(data_host) );
  
}

/*
#define NX 64
#define NY 64
#define NZ 128

hipfftHandle plan;
hipfftComplex *data1, *data2;
hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY*NZ);
hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY*NZ);
// Create a 3D FFT plan. 
hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

// Transform the first signal in place.
hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

// Transform the second signal using the same plan.
hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

// Destroy the cuFFT plan.
hipfftDestroy(plan);
hipFree(data1); hipFree(data2);
*/