#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#define N ((uint64_t) 32)

#define M_2PI (6.283185307179586)
#define SQRT_2PI (2.5066282746310002)
#define INV_SQRT_2PI (0.3989422804014327)
#define SIGMA (1)

/*
 * compile: 	nvcc -dc -lm -g -G -Xptxas="-v" -m64 -O3 -o simple_example.o -c simple_example.cu
 * 		nvcc -m64 -arch=sm_35 -o simple_example simple_example.o -lcufft_static -lculibos
 * http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/
 * + example in cuda samples
 */

/* CALLBACK TYPES
typedef enum hipfftXtCallbackType_t {
    HIPFFT_CB_LD_COMPLEX = 0x0,
    HIPFFT_CB_LD_COMPLEX_DOUBLE = 0x1,
    HIPFFT_CB_LD_REAL = 0x2,
    HIPFFT_CB_LD_REAL_DOUBLE = 0x3,
    HIPFFT_CB_ST_COMPLEX = 0x4,
    HIPFFT_CB_ST_COMPLEX_DOUBLE = 0x5,
    HIPFFT_CB_ST_REAL = 0x6,
    HIPFFT_CB_ST_REAL_DOUBLE = 0x7,
    HIPFFT_CB_UNDEFINED = 0x8
} hipfftXtCallbackType;


Read more at: http://docs.nvidia.com/cuda/cufft/index.html#ixzz3fPgy3600
Follow us: @GPUComputing on Twitter | NVIDIA on Facebook

*/


static __device__ hipfftDoubleComplex cufftSgn(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
    if (offset < N/2) {
      //((hipfftDoubleComplex* ) dataIn)[offset] = make_hipDoubleComplex(-1.,0.);
      printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(-1.,0.);
    }
    else {
      
      printf("index: %lu\tvalue:%f",-1);
      return make_hipDoubleComplex(1.,0.);
    }
}

static __device__ hipfftDoubleComplex cudaGauss_1d(void *dataIn, 
					      size_t offset, 
					      void *callerInfo, 
					      void *sharedPtr) 
{
  // get the index of thread
  //uint64_t ii = offset;
  
  
  // allocate constants in shared memory
  const double x0 = (-5*SIGMA);
  const double dx = (10*SIGMA)/((double) N);
  
  //if (ii < N) {
  //  ((hipfftDoubleComplex)* dataIn)[ii] = make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + ii*dx)*(x0 + ii*dx)/2/SIGMA)/SIGMA, 0. );
    
  //}
  return make_hipDoubleComplex( INV_SQRT_2PI*exp(-(x0 + offset*dx)*(x0 + offset*dx)/2/SIGMA)/SIGMA, 0. );
}


// pointer to callback function (on device)
__device__ hipfftCallbackLoadZ d_loadCallbackPtr = cudaGauss_1d;


int main (){
  
  hipDeviceReset();
  hipDeviceSynchronize();
  
  const uint8_t filename_str_lenght = 128;
  const uint8_t dim = 1;
    
  char filename1d[filename_str_lenght];
  FILE *file1d;
  
  sprintf(filename1d,"cufft_%dd_N%lu.bin",dim,N );
  printf("1d cufft example save in: %s\n",filename1d);
  file1d = fopen(filename1d, "wb");
  if (file1d == NULL)
  {
      printf("Error opening file %s!\n",filename1d);
      exit(EXIT_FAILURE);
  }
  
  printf("N %lu\n",N);
  
  hipfftDoubleComplex *data_dev;
  hipMalloc((void**)&data_dev, sizeof(hipfftDoubleComplex)*N);
  hipDeviceSynchronize();
  
  hipfftDoubleComplex* data_host;
  hipHostAlloc((void**) &data_host, sizeof(hipfftDoubleComplex)*N, hipHostMallocDefault);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    exit(EXIT_FAILURE);
  }
  hipDeviceSynchronize();
  
  
  // get host-usable pointer to callback functions
  hipfftCallbackLoadZ h_loadCallbackPtr;
  hipMemcpyFromSymbol(&h_loadCallbackPtr, 
                       HIP_SYMBOL(d_loadCallbackPtr), 
                       sizeof(h_loadCallbackPtr) );
  hipDeviceSynchronize();
  
  /*hipfftCallbackStoreC h_storeCallbackPtr;
  hipMemcpyFromSymbol(&h_storeCallbackPtr, 
                       HIP_SYMBOL(d_storeCallbackPtr), 
                       sizeof(h_storeCallbackPtr));*/
  
  
  // creating plan with callback
  hipfftHandle plan;
  hipfftCreate(&plan);
  
  size_t work_size;
  hipfftMakePlan1d(plan, N, HIPFFT_Z2Z, 1, &work_size); //hipfftMakePlan1d(hipfftHandle *plan, int nx, hipfftType type, int batch)
  
  /*if (hipfftPlan1d(&plan, N, HIPFFT_C2C, 1) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(EXIT_FAILURE);
  }*/
  hipDeviceSynchronize();
  hipfftResult status = hipfftXtSetCallback(plan,
		     (void **) &h_loadCallbackPtr,
                     HIPFFT_CB_LD_COMPLEX_DOUBLE,
                     NULL ); //<- here can be added structure with data needed for callback execution!
  if (status == CUFFT_LICENSE_ERROR)
  {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
  }
  
  hipDeviceSynchronize();
  
  // data will be generated in load callback!
  if (hipfftExecZ2Z(plan, data_dev, data_dev, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    exit(EXIT_FAILURE);
  }
  hipDeviceSynchronize();
  
  hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("fft data forward:\n");
  for (uint64_t ii = 0; ii < N; ii++) {
    printf("%lf + %lfj\n", hipCreal(data_host[ii]), hipCimag(data_host[ii]));
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  
  if (hipfftExecZ2Z(plan, data_dev, data_dev, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
    exit(EXIT_FAILURE);;
  }
  hipDeviceSynchronize();
  
  hipMemcpy(data_host, data_dev, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("fft data backward:\n");
  for (uint64_t ii = 0; ii < N; ii++) {
    printf("%lf + %lfj\n", hipCreal(data_host[ii]), hipCimag(data_host[ii]));
    fwrite(data_host+ii, sizeof(hipDoubleComplex),1,file1d);
  }
  
  
  fclose(file1d);
  
  hipFree(data_dev);
  hipHostFree(data_host);
  hipDeviceSynchronize();
  
  hipDeviceReset();
  hipDeviceSynchronize();
  
  return EXIT_SUCCESS;
}