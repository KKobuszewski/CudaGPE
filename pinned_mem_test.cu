
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

int main(){
  
  uint64_t N = 1<<26;
  printf("N=%lu\n",N);
  
  hipDeviceReset();
  
  hipDoubleComplex* data_host;
  hipDoubleComplex* data_dev1;
  hipDoubleComplex* data_dev2;
  hipDoubleComplex* data_dev3;
  hipDoubleComplex* data_dev4;
  printf("sizeof hipDoubleComplex %lu\n",sizeof(hipDoubleComplex));
  printf("sizeof memory %lu\n",sizeof(hipDoubleComplex)*N);
  hipHostAlloc((void**) &data_host, sizeof(hipDoubleComplex)*N, hipHostMallocDefault);
  
  
  hipMalloc((void**) &data_dev1, sizeof(hipDoubleComplex)*N);
  printf("1st array\n");
  /*cudaMemcpyAsync 	( 	void *  	dst,
		const void *  	src,
sizeof(cuDoubleComplex)*N,)*/
  hipMalloc((void**) &data_dev2, sizeof(hipDoubleComplex)*N);
  printf("2nd array\n");
  hipMalloc((void**) &data_dev3, sizeof(hipDoubleComplex)*N);
  printf("3rd array\n");
  hipMalloc((void**) &data_dev4, sizeof(hipDoubleComplex)*N);
  printf("4th array\n");
  
   	
  
  
  hipHostFree(data_host);
  hipFree(data_dev1);
  hipFree(data_dev2);
  hipFree(data_dev3);
  hipFree(data_dev4);
  
  return EXIT_SUCCESS;
}